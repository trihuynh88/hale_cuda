#include "hip/hip_runtime.h"
#include <iostream>
#include <Hale.h>
#include <glm/glm.hpp>

#include "unistd.h" // for sleep()

#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "lib/Image.h"

//from cuda_volume_rendering
#define PI 3.14159265


texture<float, 3, hipReadModeElementType> tex0;  // 3D texture
texture<float, 3, hipReadModeElementType> tex1;  // 3D texture
hipArray *d_volumeArray0 = 0;
hipArray *d_volumeArray1 = 0;

// w0, w1, w2, and w3 are the four cubic B-spline basis functions
__host__ __device__
float w0(float a)
{
    return (1.0f/6.0f)*(a*(a*(-a + 3.0f) - 3.0f) + 1.0f);
}

__host__ __device__
float w1(float a)
{
    return (1.0f/6.0f)*(a*a*(3.0f*a - 6.0f) + 4.0f);
}

__host__ __device__
float w2(float a)
{
    return (1.0f/6.0f)*(a*(a*(-3.0f*a + 3.0f) + 3.0f) + 1.0f);
}

__host__ __device__
float w3(float a)
{
    return (1.0f/6.0f)*(a*a*a);
}

//derivatives of basic functions
__host__ __device__
float w0g(float a)
{
    return -(1.0f/2.0f)*a*a + a - (1.0f/2.0f);
}

__host__ __device__
float w1g(float a)
{

    return (3.0f/2.0f)*a*a - 2*a;
}

__host__ __device__
float w2g(float a)
{
    return -(3.0f/2.0f)*a*a + a + (1.0/2.0);
}

__host__ __device__
float w3g(float a)
{
    return (1.0f/2.0f)*a*a;
}

//second derivatives of basic functions
__host__ __device__
float w0gg(float a)
{
    return 1-a;
}

__host__ __device__
float w1gg(float a)
{

    return 3*a-2;
}

__host__ __device__
float w2gg(float a)
{
    return 1-3*a;
}

__host__ __device__
float w3gg(float a)
{
    return a;
}



// filter 4 values using cubic splines
template<class T>
__device__
T cubicFilter(float x, T c0, T c1, T c2, T c3)
{
    T r;
    r = c0 * w0(x);
    r += c1 * w1(x);
    r += c2 * w2(x);
    r += c3 * w3(x);
    return r;
}

//filtering with derivative of basic functions
template<class T>
__device__
T cubicFilter_G(float x, T c0, T c1, T c2, T c3)
{
    T r;
    r = c0 * w0g(x);
    r += c1 * w1g(x);
    r += c2 * w2g(x);
    r += c3 * w3g(x);
    return r;
}

//filtering with second derivative of basic functions
template<class T>
__device__
T cubicFilter_GG(float x, T c0, T c1, T c2, T c3)
{
    T r;
    r = c0 * w0gg(x);
    r += c1 * w1gg(x);
    r += c2 * w2gg(x);
    r += c3 * w3gg(x);
    return r;
}


template<class T, class R>  // texture data type, return type
__device__
R tex3DBicubicXY(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float px = floor(x);
    float py = floor(y);
    float fx = x - px;
    float fy = y - py;

    return cubicFilter<R>(fy,
                          cubicFilter<R>(fx, tex3D(texref, px-1, py-1,z), tex3D(texref, px, py-1,z), tex3D(texref, px+1, py-1,z), tex3D(texref, px+2,py-1,z)),
                          cubicFilter<R>(fx, tex3D(texref, px-1, py,z),   tex3D(texref, px, py,z),   tex3D(texref, px+1, py,z),   tex3D(texref, px+2, py,z)),
                          cubicFilter<R>(fx, tex3D(texref, px-1, py+1,z), tex3D(texref, px, py+1,z), tex3D(texref, px+1, py+1,z), tex3D(texref, px+2, py+1,z)),
                          cubicFilter<R>(fx, tex3D(texref, px-1, py+2,z), tex3D(texref, px, py+2,z), tex3D(texref, px+1, py+2,z), tex3D(texref, px+2, py+2,z))
                         );
}

//gradient in X direction
template<class T, class R>  // texture data type, return type
__device__
R tex3DBicubicXY_GX(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float px = floor(x);
    float py = floor(y);
    float fx = x - px;
    float fy = y - py;

    return cubicFilter<R>(fy,
                          cubicFilter_G<R>(fx, tex3D(texref, px-1, py-1,z), tex3D(texref, px, py-1,z), tex3D(texref, px+1, py-1,z), tex3D(texref, px+2,py-1,z)),
                          cubicFilter_G<R>(fx, tex3D(texref, px-1, py,z),   tex3D(texref, px, py,z),   tex3D(texref, px+1, py,z),   tex3D(texref, px+2, py,z)),
                          cubicFilter_G<R>(fx, tex3D(texref, px-1, py+1,z), tex3D(texref, px, py+1,z), tex3D(texref, px+1, py+1,z), tex3D(texref, px+2, py+1,z)),
                          cubicFilter_G<R>(fx, tex3D(texref, px-1, py+2,z), tex3D(texref, px, py+2,z), tex3D(texref, px+1, py+2,z), tex3D(texref, px+2, py+2,z))
                         );
}

template<class T, class R>  // texture data type, return type
__device__
R tex3DBicubicXY_GY(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float px = floor(x);
    float py = floor(y);
    float fx = x - px;
    float fy = y - py;

    return cubicFilter_G<R>(fy,
                          cubicFilter<R>(fx, tex3D(texref, px-1, py-1,z), tex3D(texref, px, py-1,z), tex3D(texref, px+1, py-1,z), tex3D(texref, px+2,py-1,z)),
                          cubicFilter<R>(fx, tex3D(texref, px-1, py,z),   tex3D(texref, px, py,z),   tex3D(texref, px+1, py,z),   tex3D(texref, px+2, py,z)),
                          cubicFilter<R>(fx, tex3D(texref, px-1, py+1,z), tex3D(texref, px, py+1,z), tex3D(texref, px+1, py+1,z), tex3D(texref, px+2, py+1,z)),
                          cubicFilter<R>(fx, tex3D(texref, px-1, py+2,z), tex3D(texref, px, py+2,z), tex3D(texref, px+1, py+2,z), tex3D(texref, px+2, py+2,z))
                         );
}

template<class T, class R>
__device__
R tex3DBicubic(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float pz = floor(z);
    float fz = z - pz;
    return cubicFilter<R>(fz,
                          tex3DBicubicXY<T,R>(texref,x,y,pz-1),
                          tex3DBicubicXY<T,R>(texref,x,y,pz),
                          tex3DBicubicXY<T,R>(texref,x,y,pz+1),
                          tex3DBicubicXY<T,R>(texref,x,y,pz+2)
                          );
}

template<class T, class R>
__device__
R tex3DBicubic_GX(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float pz = floor(z);
    float fz = z - pz;
    return cubicFilter<R>(fz,
                          tex3DBicubicXY_GX<T,R>(texref,x,y,pz-1),
                          tex3DBicubicXY_GX<T,R>(texref,x,y,pz),
                          tex3DBicubicXY_GX<T,R>(texref,x,y,pz+1),
                          tex3DBicubicXY_GX<T,R>(texref,x,y,pz+2)
                          );
}

template<class T, class R>
__device__
R tex3DBicubic_GY(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float pz = floor(z);
    float fz = z - pz;
    return cubicFilter<R>(fz,
                          tex3DBicubicXY_GY<T,R>(texref,x,y,pz-1),
                          tex3DBicubicXY_GY<T,R>(texref,x,y,pz),
                          tex3DBicubicXY_GY<T,R>(texref,x,y,pz+1),
                          tex3DBicubicXY_GY<T,R>(texref,x,y,pz+2)
                          );
}

template<class T, class R>
__device__
R tex3DBicubic_GZ(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float pz = floor(z);
    float fz = z - pz;
    return cubicFilter_G<R>(fz,
                            tex3DBicubicXY<T,R>(texref,x,y,pz-1),
                            tex3DBicubicXY<T,R>(texref,x,y,pz),
                            tex3DBicubicXY<T,R>(texref,x,y,pz+1),
                            tex3DBicubicXY<T,R>(texref,x,y,pz+2)
                            );
}

template<class T, class R>  // texture data type, return type
__device__
R tex3DBicubicXY_GGX(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float px = floor(x);
    float py = floor(y);
    float fx = x - px;
    float fy = y - py;

    return cubicFilter<R>(fy,
                          cubicFilter_GG<R>(fx, tex3D(texref, px-1, py-1,z), tex3D(texref, px, py-1,z), tex3D(texref, px+1, py-1,z), tex3D(texref, px+2,py-1,z)),
                          cubicFilter_GG<R>(fx, tex3D(texref, px-1, py,z),   tex3D(texref, px, py,z),   tex3D(texref, px+1, py,z),   tex3D(texref, px+2, py,z)),
                          cubicFilter_GG<R>(fx, tex3D(texref, px-1, py+1,z), tex3D(texref, px, py+1,z), tex3D(texref, px+1, py+1,z), tex3D(texref, px+2, py+1,z)),
                          cubicFilter_GG<R>(fx, tex3D(texref, px-1, py+2,z), tex3D(texref, px, py+2,z), tex3D(texref, px+1, py+2,z), tex3D(texref, px+2, py+2,z))
                         );
}

template<class T, class R>  // texture data type, return type
__device__
R tex3DBicubicXY_GGY(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float px = floor(x);
    float py = floor(y);
    float fx = x - px;
    float fy = y - py;

    return cubicFilter_GG<R>(fy,
                          cubicFilter<R>(fx, tex3D(texref, px-1, py-1,z), tex3D(texref, px, py-1,z), tex3D(texref, px+1, py-1,z), tex3D(texref, px+2,py-1,z)),
                          cubicFilter<R>(fx, tex3D(texref, px-1, py,z),   tex3D(texref, px, py,z),   tex3D(texref, px+1, py,z),   tex3D(texref, px+2, py,z)),
                          cubicFilter<R>(fx, tex3D(texref, px-1, py+1,z), tex3D(texref, px, py+1,z), tex3D(texref, px+1, py+1,z), tex3D(texref, px+2, py+1,z)),
                          cubicFilter<R>(fx, tex3D(texref, px-1, py+2,z), tex3D(texref, px, py+2,z), tex3D(texref, px+1, py+2,z), tex3D(texref, px+2, py+2,z))
                         );
}

//derivative through X, then through Y
template<class T, class R>  // texture data type, return type
__device__
R tex3DBicubicXY_GYGX(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float px = floor(x);
    float py = floor(y);
    float fx = x - px;
    float fy = y - py;

    return cubicFilter_G<R>(fy,
                          cubicFilter_G<R>(fx, tex3D(texref, px-1, py-1,z), tex3D(texref, px, py-1,z), tex3D(texref, px+1, py-1,z), tex3D(texref, px+2,py-1,z)),
                          cubicFilter_G<R>(fx, tex3D(texref, px-1, py,z),   tex3D(texref, px, py,z),   tex3D(texref, px+1, py,z),   tex3D(texref, px+2, py,z)),
                          cubicFilter_G<R>(fx, tex3D(texref, px-1, py+1,z), tex3D(texref, px, py+1,z), tex3D(texref, px+1, py+1,z), tex3D(texref, px+2, py+1,z)),
                          cubicFilter_G<R>(fx, tex3D(texref, px-1, py+2,z), tex3D(texref, px, py+2,z), tex3D(texref, px+1, py+2,z), tex3D(texref, px+2, py+2,z))
                         );
}

template<class T, class R>
__device__
R tex3DBicubic_GGX(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float pz = floor(z);
    float fz = z - pz;
    return cubicFilter<R>(fz,
                          tex3DBicubicXY_GGX<T,R>(texref,x,y,pz-1),
                          tex3DBicubicXY_GGX<T,R>(texref,x,y,pz),
                          tex3DBicubicXY_GGX<T,R>(texref,x,y,pz+1),
                          tex3DBicubicXY_GGX<T,R>(texref,x,y,pz+2)
                          );
}

template<class T, class R>
__device__
R tex3DBicubic_GGY(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float pz = floor(z);
    float fz = z - pz;
    return cubicFilter<R>(fz,
                          tex3DBicubicXY_GGY<T,R>(texref,x,y,pz-1),
                          tex3DBicubicXY_GGY<T,R>(texref,x,y,pz),
                          tex3DBicubicXY_GGY<T,R>(texref,x,y,pz+1),
                          tex3DBicubicXY_GGY<T,R>(texref,x,y,pz+2)
                          );
}

template<class T, class R>
__device__
R tex3DBicubic_GGZ(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float pz = floor(z);
    float fz = z - pz;
    return cubicFilter_GG<R>(fz,
                            tex3DBicubicXY<T,R>(texref,x,y,pz-1),
                            tex3DBicubicXY<T,R>(texref,x,y,pz),
                            tex3DBicubicXY<T,R>(texref,x,y,pz+1),
                            tex3DBicubicXY<T,R>(texref,x,y,pz+2)
                            );
}

//derivative through X, then through Y
template<class T, class R>
__device__
R tex3DBicubic_GYGX(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float pz = floor(z);
    float fz = z - pz;
    return cubicFilter<R>(fz,
                          tex3DBicubicXY_GYGX<T,R>(texref,x,y,pz-1),
                          tex3DBicubicXY_GYGX<T,R>(texref,x,y,pz),
                          tex3DBicubicXY_GYGX<T,R>(texref,x,y,pz+1),
                          tex3DBicubicXY_GYGX<T,R>(texref,x,y,pz+2)
                          );
}

//derivative through X, then through Z
template<class T, class R>
__device__
R tex3DBicubic_GZGX(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float pz = floor(z);
    float fz = z - pz;
    return cubicFilter_G<R>(fz,
                          tex3DBicubicXY_GX<T,R>(texref,x,y,pz-1),
                          tex3DBicubicXY_GX<T,R>(texref,x,y,pz),
                          tex3DBicubicXY_GX<T,R>(texref,x,y,pz+1),
                          tex3DBicubicXY_GX<T,R>(texref,x,y,pz+2)
                          );
}

//derivative through Y, then through Z
template<class T, class R>
__device__
R tex3DBicubic_GZGY(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float pz = floor(z);
    float fz = z - pz;
    return cubicFilter_G<R>(fz,
                          tex3DBicubicXY_GY<T,R>(texref,x,y,pz-1),
                          tex3DBicubicXY_GY<T,R>(texref,x,y,pz),
                          tex3DBicubicXY_GY<T,R>(texref,x,y,pz+1),
                          tex3DBicubicXY_GY<T,R>(texref,x,y,pz+2)
                          );
}


__host__ __device__
int cu_getIndex2(int i, int j, int s1, int s2)
{
    return i*s2+j;
}

__host__ __device__
double dotProduct(double *u, double *v, int s)
{
    double result = 0;
    for (int i=0; i<s; i++)
        result += (u[i]*v[i]);
    return result;
}

__host__ __device__
double lenVec(double *a, int s)
{
    double len = 0;
    for (int i=0; i<s; i++)
        len += (a[i]*a[i]);
    len = sqrt(len);
    return len;
}

__host__ __device__
void addVector(double *a, double *b, double *c, int len)
{
  for (int i=0; i<len; i++)
    c[i] = a[i]+b[i];
}

__host__ __device__
void scaleVector(double *a, int len, double scale)
{
  for (int i=0; i<len; i++)
    a[i]*=scale;
}

void mulMatPoint(double X[4][4], double Y[4], double Z[4])
{
    for (int i=0; i<4; i++)
        Z[i] = 0;

    for (int i=0; i<4; i++)
        for (int k=0; k<4; k++)
            Z[i] += (X[i][k]*Y[k]);
}


__device__
void cu_mulMatPoint(double* X, double* Y, double* Z)
{
    for (int i=0; i<4; i++)
        Z[i] = 0;

    for (int i=0; i<4; i++)
        for (int k=0; k<4; k++)
            Z[i] += (X[cu_getIndex2(i,k,4,4)]*Y[k]);
}

__device__
void cu_mulMatPoint3(double* X, double* Y, double* Z)
{
    for (int i=0; i<3; i++)
        Z[i] = 0;

    for (int i=0; i<3; i++)
        for (int k=0; k<3; k++)
            Z[i] += (X[cu_getIndex2(i,k,3,3)]*Y[k]);
}

__host__ __device__
void advancePoint(double* point, double* dir, double scale, double* newpos)
{
    for (int i=0; i<3; i++)
        newpos[i] = point[i]+dir[i]*scale;
}

__device__
bool cu_isInsideDouble(double i, double j, double k, int dim1, int dim2, int dim3)
{
    return ((i>=0)&&(i<=(dim1-1))&&(j>=0)&&(j<=(dim2-1))&&(k>=0)&&(k<=(dim3-1)));
}

__device__
double cu_computeAlpha(double val, double grad_len, double isoval, double alphamax, double thickness)
{
    if ((grad_len == 0.0) && (val == isoval))
        return alphamax;
    else
        if ((grad_len>0.0) && (isoval >= (val-thickness*grad_len)) && (isoval <= (val+thickness*grad_len)))
            return alphamax*(1-abs(isoval-val)/(grad_len*thickness));
        else
            return 0.0;
}

__device__
double cu_inAlpha(double val, double grad_len, double isoval, double thickness)
{
    if (val >= isoval)
        return 1.0;
    else
    {
        return max(0.0,(1-abs(isoval-val)/(grad_len*thickness)));
    }
}

__device__
double cu_inAlphaX(double dis, double thickness)
{
    if (dis<0)
        return 1.0;
    return max(0.0,min(1.0,1.4-fabs(dis)/thickness));
}

__host__ __device__
void normalize(double *a, int s)
{
    double len = lenVec(a,s);
    for (int i=0; i<s; i++)
        a[i] = a[i]/len;
}

__host__ __device__
double diss2P(double x1,double y1,double z1,double x2,double y2,double z2)
{
    double dis1 = x2-x1;
    double dis2 = y2-y1;
    double dis3 = z2-z1;
    return (dis1*dis1+dis2*dis2+dis3*dis3);
}

__host__ __device__
void mulMat3(double* X, double* Y, double* Z)
{
    for (int i=0; i<3; i++)
        for (int j=0; j<3; j++)
        {
            for (int k=0; k<3; k++)
            {
                Z[cu_getIndex2(i,j,3,3)] += (X[cu_getIndex2(i,k,3,3)]*Y[cu_getIndex2(k,j,3,3)]);
            }
        }
}

__host__ __device__
void invertMat33(double X[][3], double Y[][3])
{
    double det = X[0][0]* (X[1][1]* X[2][2]- X[2][1]* X[1][2])-
        X[0][1]* (X[1][0]* X[2][2]- X[1][2]* X[2][0])+
        X[0][2]* (X[1][0]* X[2][1]- X[1][1]* X[2][0]);

    double invdet = 1 / det;

    Y[0][0]= (X[1][1]* X[2][2]- X[2][1]* X[1][2]) * invdet;
    Y[0][1]= (X[0][2]* X[2][1]- X[0][1]* X[2][2]) * invdet;
    Y[0][2]= (X[0][1]* X[1][2]- X[0][2]* X[1][1])* invdet;
    Y[1][0]= (X[1][2]* X[2][0]- X[1][0]* X[2][2])* invdet;
    Y[1][1]= (X[0][0]* X[2][2]- X[0][2]* X[2][0])* invdet;
    Y[1][2]= (X[1][0]* X[0][2]- X[0][0]* X[1][2])* invdet;
    Y[2][0]= (X[1][0]* X[2][1]- X[2][0]* X[1][1])* invdet;
    Y[2][1]= (X[2][0]* X[0][1]- X[0][0]* X[2][1])* invdet;
    Y[2][2]= (X[0][0]* X[1][1]- X[1][0]* X[0][1]) * invdet;
}

__device__
void eigenOfHess(double* hessian, double *eigval)
{
  double Dxx = hessian[cu_getIndex2(0,0,3,3)];
  double Dyy = hessian[cu_getIndex2(1,1,3,3)];
  double Dzz = hessian[cu_getIndex2(2,2,3,3)];
  double Dxy = hessian[cu_getIndex2(0,1,3,3)];
  double Dxz = hessian[cu_getIndex2(0,2,3,3)];
  double Dyz = hessian[cu_getIndex2(1,2,3,3)];

  double J1 = Dxx + Dyy + Dzz;
  double J2 = Dxx*Dyy + Dxx*Dzz + Dyy*Dzz - Dxy*Dxy - Dxz*Dxz - Dyz*Dyz;
  double J3 = 2*Dxy*Dxz*Dyz + Dxx*Dyy*Dzz - Dxz*Dxz*Dyy - Dxx*Dyz*Dyz - Dxy*Dxy*Dzz;
  double Q = (J1*J1-3*J2)/9;
  double R = (-9*J1*J2+27*J3+2*J1*J1*J1)/54;
  double theta = (1.0/3.0)*acos(R/sqrt(Q*Q*Q));
  double sqrtQ = sqrt(Q);
  double twosqrtQ = 2*sqrtQ;
  double J1o3 = J1/3;
  eigval[0] = J1o3 + twosqrtQ*cos(theta);
  eigval[1] = J1o3 + twosqrtQ*cos(theta-2*M_PI/3);
  eigval[2] = J1o3 + twosqrtQ*cos(theta+2*M_PI/3);
}

__device__
void computeHessian(double *hessian, double *p)
{
  hessian[cu_getIndex2(0,0,3,3)]=tex3DBicubic_GGX<float,float>(tex0,p[0],p[1],p[2]);
  hessian[cu_getIndex2(0,1,3,3)]=tex3DBicubic_GYGX<float,float>(tex0,p[0],p[1],p[2]);
  hessian[cu_getIndex2(0,2,3,3)]=tex3DBicubic_GZGX<float,float>(tex0,p[0],p[1],p[2]);
  hessian[cu_getIndex2(1,1,3,3)]=tex3DBicubic_GGY<float,float>(tex0,p[0],p[1],p[2]);
  hessian[cu_getIndex2(1,2,3,3)]=tex3DBicubic_GZGY<float,float>(tex0,p[0],p[1],p[2]);
  hessian[cu_getIndex2(2,2,3,3)]=tex3DBicubic_GGZ<float,float>(tex0,p[0],p[1],p[2]);

  hessian[cu_getIndex2(1,0,3,3)] = hessian[cu_getIndex2(0,1,3,3)];
  hessian[cu_getIndex2(2,0,3,3)] = hessian[cu_getIndex2(0,2,3,3)];
  hessian[cu_getIndex2(2,1,3,3)] = hessian[cu_getIndex2(1,2,3,3)];  
}

__global__
void kernel_cpr(int* dim, int *size, double *center, double *dir1, double *dir2, int nOutChannel, double* imageDouble
        )
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;

    if ((i>=size[0]) || (j>=size[1]))
        return;

    int ni = i-size[0]/2;
    int nj = size[1]/2 - j;
    double pointi[3];
    advancePoint(center,dir1,ni,pointi);
    advancePoint(pointi,dir2,nj,pointi);
    double val = tex3DBicubic<float,float>(tex0,pointi[0],pointi[1],pointi[2]);    
    imageDouble[j*size[0]*nOutChannel+i*nOutChannel] = val;
    for (int k=1; k<nOutChannel-1; k++)
      imageDouble[j*size[0]*nOutChannel+i*nOutChannel+k] = 0;
    imageDouble[j*size[0]*nOutChannel+i*nOutChannel+nOutChannel-1] = 1;   
}


double calDet44(double X[][4])
{
    double value = (
                    X[0][3]*X[1][2]*X[2][1]*X[3][0] - X[0][2]*X[1][3]*X[2][1]*X[3][0] - X[0][3]*X[1][1]*X[2][2]*X[3][0] + X[0][1]*X[1][3]*X[2][2]*X[3][0]+
                    X[0][2]*X[1][1]*X[2][3]*X[3][0] - X[0][1]*X[1][2]*X[2][3]*X[3][0] - X[0][3]*X[1][2]*X[2][0]*X[3][1] + X[0][2]*X[1][3]*X[2][0]*X[3][1]+
                    X[0][3]*X[1][0]*X[2][2]*X[3][1] - X[0][0]*X[1][3]*X[2][2]*X[3][1] - X[0][2]*X[1][0]*X[2][3]*X[3][1] + X[0][0]*X[1][2]*X[2][3]*X[3][1]+
                    X[0][3]*X[1][1]*X[2][0]*X[3][2] - X[0][1]*X[1][3]*X[2][0]*X[3][2] - X[0][3]*X[1][0]*X[2][1]*X[3][2] + X[0][0]*X[1][3]*X[2][1]*X[3][2]+
                    X[0][1]*X[1][0]*X[2][3]*X[3][2] - X[0][0]*X[1][1]*X[2][3]*X[3][2] - X[0][2]*X[1][1]*X[2][0]*X[3][3] + X[0][1]*X[1][2]*X[2][0]*X[3][3]+
                    X[0][2]*X[1][0]*X[2][1]*X[3][3] - X[0][0]*X[1][2]*X[2][1]*X[3][3] - X[0][1]*X[1][0]*X[2][2]*X[3][3] + X[0][0]*X[1][1]*X[2][2]*X[3][3]
                    );
    return value;
}

void invertMat44(double X[][4], double Y[][4])
{
    double det = calDet44(X);
    Y[0][0] = X[1][2]*X[2][3]*X[3][1] - X[1][3]*X[2][2]*X[3][1] + X[1][3]*X[2][1]*X[3][2] - X[1][1]*X[2][3]*X[3][2] - X[1][2]*X[2][1]*X[3][3] + X[1][1]*X[2][2]*X[3][3];
    Y[0][1] = X[0][3]*X[2][2]*X[3][1] - X[0][2]*X[2][3]*X[3][1] - X[0][3]*X[2][1]*X[3][2] + X[0][1]*X[2][3]*X[3][2] + X[0][2]*X[2][1]*X[3][3] - X[0][1]*X[2][2]*X[3][3];
    Y[0][2] = X[0][2]*X[1][3]*X[3][1] - X[0][3]*X[1][2]*X[3][1] + X[0][3]*X[1][1]*X[3][2] - X[0][1]*X[1][3]*X[3][2] - X[0][2]*X[1][1]*X[3][3] + X[0][1]*X[1][2]*X[3][3];
    Y[0][3] = X[0][3]*X[1][2]*X[2][1] - X[0][2]*X[1][3]*X[2][1] - X[0][3]*X[1][1]*X[2][2] + X[0][1]*X[1][3]*X[2][2] + X[0][2]*X[1][1]*X[2][3] - X[0][1]*X[1][2]*X[2][3];
    Y[1][0] = X[1][3]*X[2][2]*X[3][0] - X[1][2]*X[2][3]*X[3][0] - X[1][3]*X[2][0]*X[3][2] + X[1][0]*X[2][3]*X[3][2] + X[1][2]*X[2][0]*X[3][3] - X[1][0]*X[2][2]*X[3][3];
    Y[1][1] = X[0][2]*X[2][3]*X[3][0] - X[0][3]*X[2][2]*X[3][0] + X[0][3]*X[2][0]*X[3][2] - X[0][0]*X[2][3]*X[3][2] - X[0][2]*X[2][0]*X[3][3] + X[0][0]*X[2][2]*X[3][3];
    Y[1][2] = X[0][3]*X[1][2]*X[3][0] - X[0][2]*X[1][3]*X[3][0] - X[0][3]*X[1][0]*X[3][2] + X[0][0]*X[1][3]*X[3][2] + X[0][2]*X[1][0]*X[3][3] - X[0][0]*X[1][2]*X[3][3];
    Y[1][3] = X[0][2]*X[1][3]*X[2][0] - X[0][3]*X[1][2]*X[2][0] + X[0][3]*X[1][0]*X[2][2] - X[0][0]*X[1][3]*X[2][2] - X[0][2]*X[1][0]*X[2][3] + X[0][0]*X[1][2]*X[2][3];
    Y[2][0] = X[1][1]*X[2][3]*X[3][0] - X[1][3]*X[2][1]*X[3][0] + X[1][3]*X[2][0]*X[3][1] - X[1][0]*X[2][3]*X[3][1] - X[1][1]*X[2][0]*X[3][3] + X[1][0]*X[2][1]*X[3][3];
    Y[2][1] = X[0][3]*X[2][1]*X[3][0] - X[0][1]*X[2][3]*X[3][0] - X[0][3]*X[2][0]*X[3][1] + X[0][0]*X[2][3]*X[3][1] + X[0][1]*X[2][0]*X[3][3] - X[0][0]*X[2][1]*X[3][3];
    Y[2][2] = X[0][1]*X[1][3]*X[3][0] - X[0][3]*X[1][1]*X[3][0] + X[0][3]*X[1][0]*X[3][1] - X[0][0]*X[1][3]*X[3][1] - X[0][1]*X[1][0]*X[3][3] + X[0][0]*X[1][1]*X[3][3];
    Y[2][3] = X[0][3]*X[1][1]*X[2][0] - X[0][1]*X[1][3]*X[2][0] - X[0][3]*X[1][0]*X[2][1] + X[0][0]*X[1][3]*X[2][1] + X[0][1]*X[1][0]*X[2][3] - X[0][0]*X[1][1]*X[2][3];
    Y[3][0] = X[1][2]*X[2][1]*X[3][0] - X[1][1]*X[2][2]*X[3][0] - X[1][2]*X[2][0]*X[3][1] + X[1][0]*X[2][2]*X[3][1] + X[1][1]*X[2][0]*X[3][2] - X[1][0]*X[2][1]*X[3][2];
    Y[3][1] = X[0][1]*X[2][2]*X[3][0] - X[0][2]*X[2][1]*X[3][0] + X[0][2]*X[2][0]*X[3][1] - X[0][0]*X[2][2]*X[3][1] - X[0][1]*X[2][0]*X[3][2] + X[0][0]*X[2][1]*X[3][2];
    Y[3][2] = X[0][2]*X[1][1]*X[3][0] - X[0][1]*X[1][2]*X[3][0] - X[0][2]*X[1][0]*X[3][1] + X[0][0]*X[1][2]*X[3][1] + X[0][1]*X[1][0]*X[3][2] - X[0][0]*X[1][1]*X[3][2];
    Y[3][3] = X[0][1]*X[1][2]*X[2][0] - X[0][2]*X[1][1]*X[2][0] + X[0][2]*X[1][0]*X[2][1] - X[0][0]*X[1][2]*X[2][1] - X[0][1]*X[1][0]*X[2][2] + X[0][0]*X[1][1]*X[2][2];

    for (int i=0; i<4; i++)
        for (int j=0; j<4; j++)
            Y[i][j] = Y[i][j]/det;
}

void subtractVec(double *a, double *b, double *c, int s)
{
    for (int i=0; i<s; i++)
        c[i] = a[i]-b[i];
}

void cross(double *u, double *v, double *w)
{
    w[0] = u[1]*v[2]-u[2]*v[1];
    w[1] = u[2]*v[0]-u[0]*v[2];
    w[2] = u[0]*v[1]-u[1]*v[0];
}

void negateVec(double *a, int s)
{
    for (int i=0; i<s; i++)
        a[i] = -a[i];
}

//s1,s2,s3: fastest to slowest
void sliceImageDouble(double *input, int s1, int s2, int s3, double *output, int indS1)
{
    for (int i=0; i<s3; i++)
        for (int j=0; j<s2; j++)
        {
            output[i*s2+j] = input[i*s2*s1+j*s1+indS1]*input[i*s2*s1+j*s1+s1-1];
        }
}

unsigned char quantizeDouble(double val, double minVal, double maxVal)
{
    return (val-minVal)*255.0/(maxVal-minVal);
}

//3D data, fastest to slowest
void quantizeImageDouble3D(double *input, unsigned char *output, int s0, int s1, int s2)
{
    double maxVal[4];
    maxVal[0] = maxVal[1] = maxVal[2] = maxVal[3] = -(1<<15);
    double minVal[4];
    minVal[0] = minVal[1] = minVal[2] = minVal[3] = ((1<<15) - 1);

    for (int i=0; i<s2; i++)
        for (int j=0; j<s1; j++)
            for (int k=0; k<s0; k++)
            {
                if (input[i*s1*s0+j*s0+k]>maxVal[k])
                    maxVal[k] = input[i*s1*s0+j*s0+k];
                if (input[i*s1*s0+j*s0+k]<minVal[k])
                    minVal[k] = input[i*s1*s0+j*s0+k];
            }
    for (int i=0; i<4; i++)
        printf("minmax %d = [%f,%f]\n",i,minVal[i],maxVal[i]);
    for (int i=0; i<s2; i++)
        for (int j=0; j<s1; j++)
            for (int k=0; k<s0; k++)
            {
                output[i*s1*s0+j*s0+k] = quantizeDouble(input[i*s1*s0+j*s0+k],minVal[k],maxVal[k]);
            }
}

template<class T>
void quantizeImage3D(T *input, unsigned char *output, int s0, int s1, int s2)
{
    double maxVal[4];
    maxVal[0] = maxVal[1] = maxVal[2] = maxVal[3] = -(1<<15);
    double minVal[4];
    minVal[0] = minVal[1] = minVal[2] = minVal[3] = ((1<<15) - 1);

    for (int i=0; i<s2; i++)
        for (int j=0; j<s1; j++)
            for (int k=0; k<s0; k++)
            {
                if (input[i*s1*s0+j*s0+k]>maxVal[k])
                    maxVal[k] = input[i*s1*s0+j*s0+k];
                if (input[i*s1*s0+j*s0+k]<minVal[k])
                    minVal[k] = input[i*s1*s0+j*s0+k];
            }
    for (int i=0; i<4; i++)
        printf("minmax %d = [%f,%f]\n",i,minVal[i],maxVal[i]);
    for (int i=0; i<s2; i++)
        for (int j=0; j<s1; j++)
            for (int k=0; k<s0; k++)
            {
                output[i*s1*s0+j*s0+k] = quantizeDouble(input[i*s1*s0+j*s0+k],minVal[k],maxVal[k]);
            }
}

void applyMask(unsigned char *input, int s0, int s1, int s2, int *mask, unsigned char *output)
{
    for (int i=0; i<s2; i++)
        for (int j=0; j<s1; j++)
            for (int k=0; k<s0; k++)
            {
                output[i*s1*s0+j*s0+k] = input[i*s1*s0+j*s0+k]*mask[i*s1+j];
            }
}

void removeChannel(unsigned char *input, int s0, int s1, int s2, int chan, unsigned char *output)
{
    memcpy(output,input,s0*s1*s2*sizeof(unsigned char));
    for (int i=0; i<s2; i++)
        for (int j=0; j<s1; j++)
                output[i*s1*s0+j*s0+chan] = 0;            
}
//---end of cuda_volume_rendering functions

template<class T>
void setPlane(T* image, int s1, int s2, int s3, T val, int s1i)
{
  for (int i=0; i<s3; i++)
    for (int j=0; j<s2; j++)
      image[i*s2*s1+j*s1+s1i] = val;
}

void transposeMat33(double X[][3], double Y[][3])
{
    for (int i=0; i<3; i++)
        for (int j=i; j<3; j++)
        {
            Y[i][j]=X[j][i];
            Y[j][i]=X[i][j];
        }
}


float lerp(float y0, float y1, float x0, float x, float x1)
{
  float alpha = (x-x0)/(x1-x0);
  return y0*(1-alpha)+alpha*y1;
}

float linearizeDepth(float depth, float zNear, float zFar)
{
    return (2.0 * zFar * zNear) / (zFar + zNear - depth * (zFar - zNear));
}

float linearizeDepthOrtho(float depth, float zNear, float zFar)
{
    return (depth*(zFar-zNear)+zFar+zNear)/2;
}



template<class T>
void saveImage(int width, int height, int nchan, T *data, char *name)
{
    TGAImage *img = new TGAImage(width,height);
    

    unsigned char* dataQuantized = new unsigned char[height*width*nchan];
    quantizeImage3D<T>(data,dataQuantized,nchan,width,height);

    Colour c;    
    for(int x=0; x<height; x++)
        for(int y=0; y<width; y++)
        {
            c.a = 255;
            c.b = c.g = c.r = 0;
            switch (nchan)
            {
              case 4:
                c.a = dataQuantized[x*width*nchan+y*nchan+3];
              case 3:
                c.b = dataQuantized[x*width*nchan+y*nchan+2];
              case 2:
                c.g = dataQuantized[x*width*nchan+y*nchan+1];
              case 1:
                c.r = dataQuantized[x*width*nchan+y*nchan];
            }                                        
            img->setPixel(c,x,y);
         }
    
    img->WriteImage(name);  
    delete img;
    delete[] dataQuantized;
}

template<class T>
void saveImageWithoutQuantizing(int width, int height, int nchan, T *data, char *name)
{
    TGAImage *img = new TGAImage(width,height);
    
    Colour c;    
    for(int x=0; x<height; x++)
        for(int y=0; y<width; y++)
        {
            c.a = 255;
            c.b = c.g = c.r = 0;
            switch (nchan)
            {
              case 4:
                c.a = data[x*width*nchan+y*nchan+3];
              case 3:
                c.b = data[x*width*nchan+y*nchan+2];
              case 2:
                c.g = data[x*width*nchan+y*nchan+1];
              case 1:
                c.r = data[x*width*nchan+y*nchan];
            }                                        
            img->setPixel(c,x,y);
        }
    
    img->WriteImage(name);  
    delete img;
}

void render(Hale::Viewer *viewer){
  viewer->draw();
  viewer->bufferSwap();
}

int
main(int argc, const char **argv) {
  const char *me;
  char *err;
  hestOpt *hopt=NULL;
  hestParm *hparm;
  airArray *mop;

  char *name;
  char *texname1, *texname2;
  
  //double dir1[3]={1,0,0};
  //double dir2[3]={0,-1,0};
  double dir1[3],dir2[3];
  //double *dir1,*dir2;

  //tmp fixed track coords, and radius
  double track[3] = {366.653991263,89.6381792864,104.736646409};
  double trackhomo[4];
  trackhomo[0] = track[0];
  trackhomo[1] = track[1];
  trackhomo[2] = track[2];
  trackhomo[3] = 1;
  double trackw[4];
  double radius = 10;

//double *center;
  double center[3];
  //memcpy(center,track,sizeof(double)*3);

  int size[2];
  Nrrd *nin;
  char *outname;

  /* boilerplate hest code */
  me = argv[0];
  mop = airMopNew();
  hparm = hestParmNew();
  airMopAdd(mop, hparm, (airMopper)hestParmFree, airMopAlways);
  /* setting up the command-line options */
  hparm->respFileEnable = AIR_TRUE;
  hparm->noArgsIsNoProblem = AIR_TRUE;

  hestOptAdd(&hopt, "i", "nin", airTypeOther, 1, 1, &nin, "270.nrrd",
             "input volume to render", NULL, NULL, nrrdHestNrrd);

  hestOptAdd(&hopt, "isize", "sx sy", airTypeInt, 2, 2, size, "200 200",
             "output image sizes");

  hestOptAdd(&hopt, "dir1", "x y z", airTypeDouble, 3, 3, dir1, "1 0 0",
             "first direction of the generated image");

  hestOptAdd(&hopt, "dir2", "x y z", airTypeDouble, 3, 3, dir2, "0 -1 0",
             "second direction of the generated image");

  hestOptAdd(&hopt, "center", "x y z", airTypeDouble, 3, 3, center, "366.653991263 89.6381792864 104.736646409",
             "center of the generated image");

  hestOptAdd(&hopt, "o", "name", airTypeString, 1, 1, &outname, "cpr.tga", "name of output image");

  hestParseOrDie(hopt, argc-1, argv+1, hparm,
                 me, "demo program", AIR_TRUE, AIR_TRUE, AIR_TRUE);
  airMopAdd(mop, hopt, (airMopper)hestOptFree, airMopAlways);
  airMopAdd(mop, hopt, (airMopper)hestParseFree, airMopAlways);

  /* Compute threshold (isovalue) */

    unsigned int pixSize;
    hipChannelFormatDesc channelDesc;
    pixSize = sizeof(float);
    channelDesc = hipCreateChannelDesc<float>();

    if (3 != nin->dim && 3 != nin->spaceDim) {
        fprintf(stderr, "%s: need 3D array in 3D space, (not %uD in %uD)\n",
        argv[0], nin->dim, nin->spaceDim);
        airMopError(mop); exit(1);
    }

    double mat_trans[4][4];

    mat_trans[3][0] = mat_trans[3][1] = mat_trans[3][2] = 0;
    mat_trans[3][3] = 1;

    int dim[4];
    if (nin->dim == 3)
    {
        dim[0] = 1;
        dim[1] = nin->axis[0].size;
        dim[2] = nin->axis[1].size;
        dim[3] = nin->axis[2].size;
        for (int i=0; i<3; i++) {
            for (int j=0; j<3; j++) {
                /* for 2-channel data; this "i" should be "i+1" */
                mat_trans[j][i] = nin->axis[i].spaceDirection[j];
            }
            mat_trans[i][3] = nin->spaceOrigin[i];
        }
    }
    else //4-channel
    {
        dim[0] = nin->axis[0].size;
        dim[1] = nin->axis[1].size;
        dim[2] = nin->axis[2].size;
        dim[3] = nin->axis[3].size;
        for (int i=0; i<3; i++) {
            for (int j=0; j<3; j++) {
                /* for 2-channel data; this "i" should be "i+1" */
                mat_trans[j][i] = nin->axis[i+1].spaceDirection[j];
            }
            mat_trans[i][3] = nin->spaceOrigin[i];
        }
    }
    int channel = 1;
    //int filesize = dim[0]*dim[1]*dim[2]*dim[3]*pixSize;

    float* filemem0 = new float[dim[1]*dim[2]*dim[3]];
    float* filemem1 = new float[dim[1]*dim[2]*dim[3]];

    //filemem = (char*)nin->data;
    for (int i=0; i<dim[1]*dim[2]*dim[3]; i++)
    {
        filemem0[i] = ((short*)nin->data)[i*2];
        filemem1[i] = ((short*)nin->data)[i*2+1];
    }

    double mat_trans_inv[4][4];
    invertMat44(mat_trans,mat_trans_inv);
   //tex3D stuff
    const hipExtent volumeSize = make_hipExtent(dim[1], dim[2], dim[3]);

    //hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipMalloc3DArray(&d_volumeArray0, &channelDesc, volumeSize);
    hipMalloc3DArray(&d_volumeArray1, &channelDesc, volumeSize);

    // --- Copy data to 3D array (host to device)
    hipMemcpy3DParms copyParams1 = {0};
    copyParams1.srcPtr   = make_hipPitchedPtr((void*)filemem1, volumeSize.width*pixSize, volumeSize.width, volumeSize.height);
    copyParams1.dstArray = d_volumeArray1;
    copyParams1.extent   = volumeSize;
    copyParams1.kind     = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams1);

    hipMemcpy3DParms copyParams0 = {0};
    copyParams0.srcPtr   = make_hipPitchedPtr((void*)filemem0, volumeSize.width*pixSize, volumeSize.width, volumeSize.height);
    copyParams0.dstArray = d_volumeArray0;
    copyParams0.extent   = volumeSize;
    copyParams0.kind     = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams0);
    // --- Set texture parameters
    tex1.normalized = false;                      // access with normalized texture coordinates
    tex1.filterMode = hipFilterModeLinear;      // linear interpolation
    /*
    tex1.addressMode[0] = hipAddressModeWrap;   // wrap texture coordinates
    tex1.addressMode[1] = hipAddressModeWrap;
    tex1.addressMode[2] = hipAddressModeWrap;
    */
    tex1.addressMode[0] = hipAddressModeBorder;   // wrap texture coordinates
    tex1.addressMode[1] = hipAddressModeBorder;
    tex1.addressMode[2] = hipAddressModeBorder;


    tex0.normalized = false;                      // access with normalized texture coordinates
    tex0.filterMode = hipFilterModeLinear;      // linear interpolation
    /*
    tex0.addressMode[0] = hipAddressModeWrap;   // wrap texture coordinates
    tex0.addressMode[1] = hipAddressModeWrap;
    tex0.addressMode[2] = hipAddressModeWrap;
    */
    tex0.addressMode[0] = hipAddressModeBorder;   // wrap texture coordinates
    tex0.addressMode[1] = hipAddressModeBorder;
    tex0.addressMode[2] = hipAddressModeBorder;
    // --- Bind array to 3D texture
    hipBindTextureToArray(tex1, d_volumeArray1, channelDesc);
    hipBindTextureToArray(tex0, d_volumeArray0, channelDesc);
    //-----------

    int nOutChannel = 4;

    double *imageDouble = new double[size[0]*size[1]*nOutChannel];
    //CUDA Var

    int *d_dim;
    hipMalloc(&d_dim, sizeof(dim));
    hipMemcpy(d_dim, dim, 4*sizeof(int), hipMemcpyHostToDevice);

    double *d_dir1;
    hipMalloc(&d_dir1, sizeof(dir1));
    hipMemcpy(d_dir1, dir1, 3*sizeof(double), hipMemcpyHostToDevice);

    double *d_dir2;
    hipMalloc(&d_dir2, sizeof(dir2));
    hipMemcpy(d_dir2, dir2, 3*sizeof(double), hipMemcpyHostToDevice);

    double *d_imageDouble;
    hipMalloc(&d_imageDouble,sizeof(double)*size[0]*size[1]*nOutChannel);

    int *d_size;
    hipMalloc(&d_size,2*sizeof(int));
    hipMemcpy(d_size,size,2*sizeof(int), hipMemcpyHostToDevice);

    double *d_center;
    hipMalloc(&d_center,3*sizeof(double));
    hipMemcpy(d_center,center,3*sizeof(double), hipMemcpyHostToDevice);


    int numThread1D = 16;
    dim3 threadsPerBlock(numThread1D,numThread1D);
    dim3 numBlocks((size[0]+numThread1D-1)/numThread1D,(size[1]+numThread1D-1)/numThread1D);

    kernel_cpr<<<numBlocks,threadsPerBlock>>>(d_dim, d_size, d_center, d_dir1, d_dir2, nOutChannel, d_imageDouble);

    hipError_t errCu = hipGetLastError();
    if (errCu != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(errCu));

    errCu = hipDeviceSynchronize();
    if (errCu != hipSuccess) 
        printf("Error Sync: %s\n", hipGetErrorString(errCu));

    hipMemcpy(imageDouble, d_imageDouble, sizeof(double)*size[0]*size[1]*nOutChannel, hipMemcpyDeviceToHost);

    short width = size[0];
    short height = size[1];

    unsigned char *imageQuantized = new unsigned char[size[0]*size[1]*4];
    quantizeImageDouble3D(imageDouble,imageQuantized,4,size[0],size[1]);
    setPlane<unsigned char>(imageQuantized, 4, size[0], size[1], 255, 3);
//end of cuda_rendering

  saveImageWithoutQuantizing<unsigned char>(size[0],size[1],4,imageQuantized,outname);

  airMopOkay(mop);

  return 0;
}
