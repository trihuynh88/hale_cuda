#include "hip/hip_runtime.h"
#include <iostream>
#include <Hale.h>
#include <glm/glm.hpp>

#include "unistd.h" // for sleep()

#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "lib/Image.h"
#include <vector>

using namespace std;

//from cuda_volume_rendering
#define PI 3.14159265

#define MAX(a,b) ((a)>(b)?(a):(b))

texture<float, 3, hipReadModeElementType> tex0;  // 3D texture
texture<float, 3, hipReadModeElementType> tex1;  // 3D texture
hipArray *d_volumeArray0 = 0;
hipArray *d_volumeArray1 = 0;

// w0, w1, w2, and w3 are the four cubic B-spline basis functions
__host__ __device__
float w0(float a)
{
    return (1.0f/6.0f)*(a*(a*(-a + 3.0f) - 3.0f) + 1.0f);
}

__host__ __device__
float w1(float a)
{
    return (1.0f/6.0f)*(a*a*(3.0f*a - 6.0f) + 4.0f);
}

__host__ __device__
float w2(float a)
{
    return (1.0f/6.0f)*(a*(a*(-3.0f*a + 3.0f) + 3.0f) + 1.0f);
}

__host__ __device__
float w3(float a)
{
    return (1.0f/6.0f)*(a*a*a);
}

//derivatives of basic functions
__host__ __device__
float w0g(float a)
{
    return -(1.0f/2.0f)*a*a + a - (1.0f/2.0f);
}

__host__ __device__
float w1g(float a)
{

    return (3.0f/2.0f)*a*a - 2*a;
}

__host__ __device__
float w2g(float a)
{
    return -(3.0f/2.0f)*a*a + a + (1.0/2.0);
}

__host__ __device__
float w3g(float a)
{
    return (1.0f/2.0f)*a*a;
}

//second derivatives of basic functions
__host__ __device__
float w0gg(float a)
{
    return 1-a;
}

__host__ __device__
float w1gg(float a)
{

    return 3*a-2;
}

__host__ __device__
float w2gg(float a)
{
    return 1-3*a;
}

__host__ __device__
float w3gg(float a)
{
    return a;
}



// filter 4 values using cubic splines
template<class T>
__host__ __device__
T cubicFilter(float x, T c0, T c1, T c2, T c3)
{
    T r;
    r = c0 * w0(x);
    r += c1 * w1(x);
    r += c2 * w2(x);
    r += c3 * w3(x);
    return r;
}

//filtering with derivative of basic functions
template<class T>
__host__ __device__
T cubicFilter_G(float x, T c0, T c1, T c2, T c3)
{
    T r;
    r = c0 * w0g(x);
    r += c1 * w1g(x);
    r += c2 * w2g(x);
    r += c3 * w3g(x);
    return r;
}

//filtering with second derivative of basic functions
template<class T>
__host__ __device__
T cubicFilter_GG(float x, T c0, T c1, T c2, T c3)
{
    T r;
    r = c0 * w0gg(x);
    r += c1 * w1gg(x);
    r += c2 * w2gg(x);
    r += c3 * w3gg(x);
    return r;
}


template<class T, class R>  // texture data type, return type
__device__
R tex3DBicubicXY(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float px = floor(x);
    float py = floor(y);
    float fx = x - px;
    float fy = y - py;

    return cubicFilter<R>(fy,
                          cubicFilter<R>(fx, tex3D(texref, px-1, py-1,z), tex3D(texref, px, py-1,z), tex3D(texref, px+1, py-1,z), tex3D(texref, px+2,py-1,z)),
                          cubicFilter<R>(fx, tex3D(texref, px-1, py,z),   tex3D(texref, px, py,z),   tex3D(texref, px+1, py,z),   tex3D(texref, px+2, py,z)),
                          cubicFilter<R>(fx, tex3D(texref, px-1, py+1,z), tex3D(texref, px, py+1,z), tex3D(texref, px+1, py+1,z), tex3D(texref, px+2, py+1,z)),
                          cubicFilter<R>(fx, tex3D(texref, px-1, py+2,z), tex3D(texref, px, py+2,z), tex3D(texref, px+1, py+2,z), tex3D(texref, px+2, py+2,z))
                         );
}

//gradient in X direction
template<class T, class R>  // texture data type, return type
__device__
R tex3DBicubicXY_GX(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float px = floor(x);
    float py = floor(y);
    float fx = x - px;
    float fy = y - py;

    return cubicFilter<R>(fy,
                          cubicFilter_G<R>(fx, tex3D(texref, px-1, py-1,z), tex3D(texref, px, py-1,z), tex3D(texref, px+1, py-1,z), tex3D(texref, px+2,py-1,z)),
                          cubicFilter_G<R>(fx, tex3D(texref, px-1, py,z),   tex3D(texref, px, py,z),   tex3D(texref, px+1, py,z),   tex3D(texref, px+2, py,z)),
                          cubicFilter_G<R>(fx, tex3D(texref, px-1, py+1,z), tex3D(texref, px, py+1,z), tex3D(texref, px+1, py+1,z), tex3D(texref, px+2, py+1,z)),
                          cubicFilter_G<R>(fx, tex3D(texref, px-1, py+2,z), tex3D(texref, px, py+2,z), tex3D(texref, px+1, py+2,z), tex3D(texref, px+2, py+2,z))
                         );
}

template<class T, class R>  // texture data type, return type
__device__
R tex3DBicubicXY_GY(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float px = floor(x);
    float py = floor(y);
    float fx = x - px;
    float fy = y - py;

    return cubicFilter_G<R>(fy,
                          cubicFilter<R>(fx, tex3D(texref, px-1, py-1,z), tex3D(texref, px, py-1,z), tex3D(texref, px+1, py-1,z), tex3D(texref, px+2,py-1,z)),
                          cubicFilter<R>(fx, tex3D(texref, px-1, py,z),   tex3D(texref, px, py,z),   tex3D(texref, px+1, py,z),   tex3D(texref, px+2, py,z)),
                          cubicFilter<R>(fx, tex3D(texref, px-1, py+1,z), tex3D(texref, px, py+1,z), tex3D(texref, px+1, py+1,z), tex3D(texref, px+2, py+1,z)),
                          cubicFilter<R>(fx, tex3D(texref, px-1, py+2,z), tex3D(texref, px, py+2,z), tex3D(texref, px+1, py+2,z), tex3D(texref, px+2, py+2,z))
                         );
}

template<class T, class R>
__device__
R tex3DBicubic(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float pz = floor(z);
    float fz = z - pz;
    return cubicFilter<R>(fz,
                          tex3DBicubicXY<T,R>(texref,x,y,pz-1),
                          tex3DBicubicXY<T,R>(texref,x,y,pz),
                          tex3DBicubicXY<T,R>(texref,x,y,pz+1),
                          tex3DBicubicXY<T,R>(texref,x,y,pz+2)
                          );
}

template<class T, class R>
__device__
R tex3DBicubic_GX(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float pz = floor(z);
    float fz = z - pz;
    return cubicFilter<R>(fz,
                          tex3DBicubicXY_GX<T,R>(texref,x,y,pz-1),
                          tex3DBicubicXY_GX<T,R>(texref,x,y,pz),
                          tex3DBicubicXY_GX<T,R>(texref,x,y,pz+1),
                          tex3DBicubicXY_GX<T,R>(texref,x,y,pz+2)
                          );
}

template<class T, class R>
__device__
R tex3DBicubic_GY(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float pz = floor(z);
    float fz = z - pz;
    return cubicFilter<R>(fz,
                          tex3DBicubicXY_GY<T,R>(texref,x,y,pz-1),
                          tex3DBicubicXY_GY<T,R>(texref,x,y,pz),
                          tex3DBicubicXY_GY<T,R>(texref,x,y,pz+1),
                          tex3DBicubicXY_GY<T,R>(texref,x,y,pz+2)
                          );
}

template<class T, class R>
__device__
R tex3DBicubic_GZ(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float pz = floor(z);
    float fz = z - pz;
    return cubicFilter_G<R>(fz,
                            tex3DBicubicXY<T,R>(texref,x,y,pz-1),
                            tex3DBicubicXY<T,R>(texref,x,y,pz),
                            tex3DBicubicXY<T,R>(texref,x,y,pz+1),
                            tex3DBicubicXY<T,R>(texref,x,y,pz+2)
                            );
}

template<class T, class R>  // texture data type, return type
__device__
R tex3DBicubicXY_GGX(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float px = floor(x);
    float py = floor(y);
    float fx = x - px;
    float fy = y - py;

    return cubicFilter<R>(fy,
                          cubicFilter_GG<R>(fx, tex3D(texref, px-1, py-1,z), tex3D(texref, px, py-1,z), tex3D(texref, px+1, py-1,z), tex3D(texref, px+2,py-1,z)),
                          cubicFilter_GG<R>(fx, tex3D(texref, px-1, py,z),   tex3D(texref, px, py,z),   tex3D(texref, px+1, py,z),   tex3D(texref, px+2, py,z)),
                          cubicFilter_GG<R>(fx, tex3D(texref, px-1, py+1,z), tex3D(texref, px, py+1,z), tex3D(texref, px+1, py+1,z), tex3D(texref, px+2, py+1,z)),
                          cubicFilter_GG<R>(fx, tex3D(texref, px-1, py+2,z), tex3D(texref, px, py+2,z), tex3D(texref, px+1, py+2,z), tex3D(texref, px+2, py+2,z))
                         );
}

template<class T, class R>  // texture data type, return type
__device__
R tex3DBicubicXY_GGY(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float px = floor(x);
    float py = floor(y);
    float fx = x - px;
    float fy = y - py;

    return cubicFilter_GG<R>(fy,
                          cubicFilter<R>(fx, tex3D(texref, px-1, py-1,z), tex3D(texref, px, py-1,z), tex3D(texref, px+1, py-1,z), tex3D(texref, px+2,py-1,z)),
                          cubicFilter<R>(fx, tex3D(texref, px-1, py,z),   tex3D(texref, px, py,z),   tex3D(texref, px+1, py,z),   tex3D(texref, px+2, py,z)),
                          cubicFilter<R>(fx, tex3D(texref, px-1, py+1,z), tex3D(texref, px, py+1,z), tex3D(texref, px+1, py+1,z), tex3D(texref, px+2, py+1,z)),
                          cubicFilter<R>(fx, tex3D(texref, px-1, py+2,z), tex3D(texref, px, py+2,z), tex3D(texref, px+1, py+2,z), tex3D(texref, px+2, py+2,z))
                         );
}

//derivative through X, then through Y
template<class T, class R>  // texture data type, return type
__device__
R tex3DBicubicXY_GYGX(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float px = floor(x);
    float py = floor(y);
    float fx = x - px;
    float fy = y - py;

    return cubicFilter_G<R>(fy,
                          cubicFilter_G<R>(fx, tex3D(texref, px-1, py-1,z), tex3D(texref, px, py-1,z), tex3D(texref, px+1, py-1,z), tex3D(texref, px+2,py-1,z)),
                          cubicFilter_G<R>(fx, tex3D(texref, px-1, py,z),   tex3D(texref, px, py,z),   tex3D(texref, px+1, py,z),   tex3D(texref, px+2, py,z)),
                          cubicFilter_G<R>(fx, tex3D(texref, px-1, py+1,z), tex3D(texref, px, py+1,z), tex3D(texref, px+1, py+1,z), tex3D(texref, px+2, py+1,z)),
                          cubicFilter_G<R>(fx, tex3D(texref, px-1, py+2,z), tex3D(texref, px, py+2,z), tex3D(texref, px+1, py+2,z), tex3D(texref, px+2, py+2,z))
                         );
}

template<class T, class R>
__device__
R tex3DBicubic_GGX(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float pz = floor(z);
    float fz = z - pz;
    return cubicFilter<R>(fz,
                          tex3DBicubicXY_GGX<T,R>(texref,x,y,pz-1),
                          tex3DBicubicXY_GGX<T,R>(texref,x,y,pz),
                          tex3DBicubicXY_GGX<T,R>(texref,x,y,pz+1),
                          tex3DBicubicXY_GGX<T,R>(texref,x,y,pz+2)
                          );
}

template<class T, class R>
__device__
R tex3DBicubic_GGY(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float pz = floor(z);
    float fz = z - pz;
    return cubicFilter<R>(fz,
                          tex3DBicubicXY_GGY<T,R>(texref,x,y,pz-1),
                          tex3DBicubicXY_GGY<T,R>(texref,x,y,pz),
                          tex3DBicubicXY_GGY<T,R>(texref,x,y,pz+1),
                          tex3DBicubicXY_GGY<T,R>(texref,x,y,pz+2)
                          );
}

template<class T, class R>
__device__
R tex3DBicubic_GGZ(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float pz = floor(z);
    float fz = z - pz;
    return cubicFilter_GG<R>(fz,
                            tex3DBicubicXY<T,R>(texref,x,y,pz-1),
                            tex3DBicubicXY<T,R>(texref,x,y,pz),
                            tex3DBicubicXY<T,R>(texref,x,y,pz+1),
                            tex3DBicubicXY<T,R>(texref,x,y,pz+2)
                            );
}

//derivative through X, then through Y
template<class T, class R>
__device__
R tex3DBicubic_GYGX(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float pz = floor(z);
    float fz = z - pz;
    return cubicFilter<R>(fz,
                          tex3DBicubicXY_GYGX<T,R>(texref,x,y,pz-1),
                          tex3DBicubicXY_GYGX<T,R>(texref,x,y,pz),
                          tex3DBicubicXY_GYGX<T,R>(texref,x,y,pz+1),
                          tex3DBicubicXY_GYGX<T,R>(texref,x,y,pz+2)
                          );
}

//derivative through X, then through Z
template<class T, class R>
__device__
R tex3DBicubic_GZGX(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float pz = floor(z);
    float fz = z - pz;
    return cubicFilter_G<R>(fz,
                          tex3DBicubicXY_GX<T,R>(texref,x,y,pz-1),
                          tex3DBicubicXY_GX<T,R>(texref,x,y,pz),
                          tex3DBicubicXY_GX<T,R>(texref,x,y,pz+1),
                          tex3DBicubicXY_GX<T,R>(texref,x,y,pz+2)
                          );
}

//derivative through Y, then through Z
template<class T, class R>
__device__
R tex3DBicubic_GZGY(const texture<T, 3, hipReadModeElementType> texref, float x, float y, float z)
{
    float pz = floor(z);
    float fz = z - pz;
    return cubicFilter_G<R>(fz,
                          tex3DBicubicXY_GY<T,R>(texref,x,y,pz-1),
                          tex3DBicubicXY_GY<T,R>(texref,x,y,pz),
                          tex3DBicubicXY_GY<T,R>(texref,x,y,pz+1),
                          tex3DBicubicXY_GY<T,R>(texref,x,y,pz+2)
                          );
}


__host__ __device__
int cu_getIndex2(int i, int j, int s1, int s2)
{
    return i*s2+j;
}

__host__ __device__
double dotProduct(double *u, double *v, int s)
{
    double result = 0;
    for (int i=0; i<s; i++)
        result += (u[i]*v[i]);
    return result;
}

__host__ __device__
double lenVec(double *a, int s)
{
    double len = 0;
    for (int i=0; i<s; i++)
        len += (a[i]*a[i]);
    len = sqrt(len);
    return len;
}

__host__ __device__
void addVector(double *a, double *b, double *c, int len)
{
  for (int i=0; i<len; i++)
    c[i] = a[i]+b[i];
}

__host__ __device__
void scaleVector(double *a, int len, double scale)
{
  for (int i=0; i<len; i++)
    a[i]*=scale;
}

void mulMatPoint(double X[4][4], double Y[4], double Z[4])
{
    for (int i=0; i<4; i++)
        Z[i] = 0;

    for (int i=0; i<4; i++)
        for (int k=0; k<4; k++)
            Z[i] += (X[i][k]*Y[k]);
}


__device__
void cu_mulMatPoint(double* X, double* Y, double* Z)
{
    for (int i=0; i<4; i++)
        Z[i] = 0;

    for (int i=0; i<4; i++)
        for (int k=0; k<4; k++)
            Z[i] += (X[cu_getIndex2(i,k,4,4)]*Y[k]);
}

__device__
void cu_mulMatPoint3(double* X, double* Y, double* Z)
{
    for (int i=0; i<3; i++)
        Z[i] = 0;

    for (int i=0; i<3; i++)
        for (int k=0; k<3; k++)
            Z[i] += (X[cu_getIndex2(i,k,3,3)]*Y[k]);
}

__host__ __device__
void advancePoint(double* point, double* dir, double scale, double* newpos)
{
    for (int i=0; i<3; i++)
        newpos[i] = point[i]+dir[i]*scale;
}

__device__
bool cu_isInsideDouble(double i, double j, double k, int dim1, int dim2, int dim3)
{
    return ((i>=0)&&(i<=(dim1-1))&&(j>=0)&&(j<=(dim2-1))&&(k>=0)&&(k<=(dim3-1)));
}

__device__
double cu_computeAlpha(double val, double grad_len, double isoval, double alphamax, double thickness)
{
    if ((grad_len == 0.0) && (val == isoval))
        return alphamax;
    else
        if ((grad_len>0.0) && (isoval >= (val-thickness*grad_len)) && (isoval <= (val+thickness*grad_len)))
            return alphamax*(1-abs(isoval-val)/(grad_len*thickness));
        else
            return 0.0;
}

__device__
double cu_inAlpha(double val, double grad_len, double isoval, double thickness)
{
    if (val >= isoval)
        return 1.0;
    else
    {
        return max(0.0,(1-abs(isoval-val)/(grad_len*thickness)));
    }
}

__device__
double cu_inAlphaX(double dis, double thickness)
{
    if (dis<0)
        return 1.0;
    return max(0.0,min(1.0,1.4-fabs(dis)/thickness));
}

__host__ __device__
void normalize(double *a, int s)
{
    double len = lenVec(a,s);
    for (int i=0; i<s; i++)
        a[i] = a[i]/len;
}

__host__ __device__
double diss2P(double x1,double y1,double z1,double x2,double y2,double z2)
{
    double dis1 = x2-x1;
    double dis2 = y2-y1;
    double dis3 = z2-z1;
    return (dis1*dis1+dis2*dis2+dis3*dis3);
}

__host__ __device__
void mulMat3(double* X, double* Y, double* Z)
{
    for (int i=0; i<3; i++)
        for (int j=0; j<3; j++)
        {
            for (int k=0; k<3; k++)
            {
                Z[cu_getIndex2(i,j,3,3)] += (X[cu_getIndex2(i,k,3,3)]*Y[cu_getIndex2(k,j,3,3)]);
            }
        }
}

__host__ __device__
void invertMat33(double X[][3], double Y[][3])
{
    double det = X[0][0]* (X[1][1]* X[2][2]- X[2][1]* X[1][2])-
        X[0][1]* (X[1][0]* X[2][2]- X[1][2]* X[2][0])+
        X[0][2]* (X[1][0]* X[2][1]- X[1][1]* X[2][0]);

    double invdet = 1 / det;

    Y[0][0]= (X[1][1]* X[2][2]- X[2][1]* X[1][2]) * invdet;
    Y[0][1]= (X[0][2]* X[2][1]- X[0][1]* X[2][2]) * invdet;
    Y[0][2]= (X[0][1]* X[1][2]- X[0][2]* X[1][1])* invdet;
    Y[1][0]= (X[1][2]* X[2][0]- X[1][0]* X[2][2])* invdet;
    Y[1][1]= (X[0][0]* X[2][2]- X[0][2]* X[2][0])* invdet;
    Y[1][2]= (X[1][0]* X[0][2]- X[0][0]* X[1][2])* invdet;
    Y[2][0]= (X[1][0]* X[2][1]- X[2][0]* X[1][1])* invdet;
    Y[2][1]= (X[2][0]* X[0][1]- X[0][0]* X[2][1])* invdet;
    Y[2][2]= (X[0][0]* X[1][1]- X[1][0]* X[0][1]) * invdet;
}

__device__
void eigenOfHess(double* hessian, double *eigval)
{
  double Dxx = hessian[cu_getIndex2(0,0,3,3)];
  double Dyy = hessian[cu_getIndex2(1,1,3,3)];
  double Dzz = hessian[cu_getIndex2(2,2,3,3)];
  double Dxy = hessian[cu_getIndex2(0,1,3,3)];
  double Dxz = hessian[cu_getIndex2(0,2,3,3)];
  double Dyz = hessian[cu_getIndex2(1,2,3,3)];

  double J1 = Dxx + Dyy + Dzz;
  double J2 = Dxx*Dyy + Dxx*Dzz + Dyy*Dzz - Dxy*Dxy - Dxz*Dxz - Dyz*Dyz;
  double J3 = 2*Dxy*Dxz*Dyz + Dxx*Dyy*Dzz - Dxz*Dxz*Dyy - Dxx*Dyz*Dyz - Dxy*Dxy*Dzz;
  double Q = (J1*J1-3*J2)/9;
  double R = (-9*J1*J2+27*J3+2*J1*J1*J1)/54;
  double theta = (1.0/3.0)*acos(R/sqrt(Q*Q*Q));
  double sqrtQ = sqrt(Q);
  double twosqrtQ = 2*sqrtQ;
  double J1o3 = J1/3;
  eigval[0] = J1o3 + twosqrtQ*cos(theta);
  eigval[1] = J1o3 + twosqrtQ*cos(theta-2*M_PI/3);
  eigval[2] = J1o3 + twosqrtQ*cos(theta+2*M_PI/3);
}

__device__
void computeHessian(double *hessian, double *p)
{
  hessian[cu_getIndex2(0,0,3,3)]=tex3DBicubic_GGX<float,float>(tex0,p[0],p[1],p[2]);
  hessian[cu_getIndex2(0,1,3,3)]=tex3DBicubic_GYGX<float,float>(tex0,p[0],p[1],p[2]);
  hessian[cu_getIndex2(0,2,3,3)]=tex3DBicubic_GZGX<float,float>(tex0,p[0],p[1],p[2]);
  hessian[cu_getIndex2(1,1,3,3)]=tex3DBicubic_GGY<float,float>(tex0,p[0],p[1],p[2]);
  hessian[cu_getIndex2(1,2,3,3)]=tex3DBicubic_GZGY<float,float>(tex0,p[0],p[1],p[2]);
  hessian[cu_getIndex2(2,2,3,3)]=tex3DBicubic_GGZ<float,float>(tex0,p[0],p[1],p[2]);

  hessian[cu_getIndex2(1,0,3,3)] = hessian[cu_getIndex2(0,1,3,3)];
  hessian[cu_getIndex2(2,0,3,3)] = hessian[cu_getIndex2(0,2,3,3)];
  hessian[cu_getIndex2(2,1,3,3)] = hessian[cu_getIndex2(1,2,3,3)];  
}

__host__ __device__
void cross(double *u, double *v, double *w)
{
    w[0] = u[1]*v[2]-u[2]*v[1];
    w[1] = u[2]*v[0]-u[0]*v[2];
    w[2] = u[0]*v[1]-u[1]*v[0];
}

//currently working in index-space
//do MIP for a small slice around each point
__global__
void kernel_cpr(int* dim, int *size, double *center, double *dir1, double *dir2, double swidth, double sstep, int nOutChannel, double* imageDouble
        )
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;

    if ((i>=size[0]) || (j>=size[1]))
        return;

    int ni = i-size[0]/2;
    int nj = size[1]/2 - j;
    double pointi[3];
    advancePoint(center,dir1,ni,pointi);
    advancePoint(pointi,dir2,nj,pointi);

    double mipdir[3];
    cross(dir1,dir2,mipdir);
    normalize(mipdir,3);

    double mipval = INT_MIN;

    double curpoint[3];
    int k;
    for (k=0; k<3; k++)
      curpoint[k] = pointi[k] - mipdir[k]*swidth/2;

    for (k=0; k<ceil(swidth/sstep); k++)
    {
      double curval;
      curval = tex3DBicubic<float,float>(tex0,curpoint[0],curpoint[1],curpoint[2]);    
      mipval = MAX(mipval,curval);
      curpoint[0] = curpoint[0] + mipdir[0]*sstep;
      curpoint[1] = curpoint[1] + mipdir[1]*sstep;
      curpoint[2] = curpoint[2] + mipdir[2]*sstep;
    }

    //double val = tex3DBicubic<float,float>(tex0,pointi[0],pointi[1],pointi[2]);    
    imageDouble[j*size[0]*nOutChannel+i*nOutChannel] = mipval;
    for (int k=1; k<nOutChannel-1; k++)
      imageDouble[j*size[0]*nOutChannel+i*nOutChannel+k] = 0;
    imageDouble[j*size[0]*nOutChannel+i*nOutChannel+nOutChannel-1] = 1;   
}


void drawCircle(unsigned char *img, int s0, int s1, int s2, int drawchan, int c1, int c2, double rad)
{
  double angstep = 0.2;
  for (double curang = 0; curang<2*M_PI; curang+=angstep)
  {
    int i1, i2;
    i2 = sin(curang)*rad;
    i1 = cos(curang)*rad;
    i1 += c1;
    i2 += c2;

    img[i2*s1*s0 + i1*s0 + drawchan] = 255;
  }
}

double calDet44(double X[][4])
{
    double value = (
                    X[0][3]*X[1][2]*X[2][1]*X[3][0] - X[0][2]*X[1][3]*X[2][1]*X[3][0] - X[0][3]*X[1][1]*X[2][2]*X[3][0] + X[0][1]*X[1][3]*X[2][2]*X[3][0]+
                    X[0][2]*X[1][1]*X[2][3]*X[3][0] - X[0][1]*X[1][2]*X[2][3]*X[3][0] - X[0][3]*X[1][2]*X[2][0]*X[3][1] + X[0][2]*X[1][3]*X[2][0]*X[3][1]+
                    X[0][3]*X[1][0]*X[2][2]*X[3][1] - X[0][0]*X[1][3]*X[2][2]*X[3][1] - X[0][2]*X[1][0]*X[2][3]*X[3][1] + X[0][0]*X[1][2]*X[2][3]*X[3][1]+
                    X[0][3]*X[1][1]*X[2][0]*X[3][2] - X[0][1]*X[1][3]*X[2][0]*X[3][2] - X[0][3]*X[1][0]*X[2][1]*X[3][2] + X[0][0]*X[1][3]*X[2][1]*X[3][2]+
                    X[0][1]*X[1][0]*X[2][3]*X[3][2] - X[0][0]*X[1][1]*X[2][3]*X[3][2] - X[0][2]*X[1][1]*X[2][0]*X[3][3] + X[0][1]*X[1][2]*X[2][0]*X[3][3]+
                    X[0][2]*X[1][0]*X[2][1]*X[3][3] - X[0][0]*X[1][2]*X[2][1]*X[3][3] - X[0][1]*X[1][0]*X[2][2]*X[3][3] + X[0][0]*X[1][1]*X[2][2]*X[3][3]
                    );
    return value;
}

void invertMat44(double X[][4], double Y[][4])
{
    double det = calDet44(X);
    Y[0][0] = X[1][2]*X[2][3]*X[3][1] - X[1][3]*X[2][2]*X[3][1] + X[1][3]*X[2][1]*X[3][2] - X[1][1]*X[2][3]*X[3][2] - X[1][2]*X[2][1]*X[3][3] + X[1][1]*X[2][2]*X[3][3];
    Y[0][1] = X[0][3]*X[2][2]*X[3][1] - X[0][2]*X[2][3]*X[3][1] - X[0][3]*X[2][1]*X[3][2] + X[0][1]*X[2][3]*X[3][2] + X[0][2]*X[2][1]*X[3][3] - X[0][1]*X[2][2]*X[3][3];
    Y[0][2] = X[0][2]*X[1][3]*X[3][1] - X[0][3]*X[1][2]*X[3][1] + X[0][3]*X[1][1]*X[3][2] - X[0][1]*X[1][3]*X[3][2] - X[0][2]*X[1][1]*X[3][3] + X[0][1]*X[1][2]*X[3][3];
    Y[0][3] = X[0][3]*X[1][2]*X[2][1] - X[0][2]*X[1][3]*X[2][1] - X[0][3]*X[1][1]*X[2][2] + X[0][1]*X[1][3]*X[2][2] + X[0][2]*X[1][1]*X[2][3] - X[0][1]*X[1][2]*X[2][3];
    Y[1][0] = X[1][3]*X[2][2]*X[3][0] - X[1][2]*X[2][3]*X[3][0] - X[1][3]*X[2][0]*X[3][2] + X[1][0]*X[2][3]*X[3][2] + X[1][2]*X[2][0]*X[3][3] - X[1][0]*X[2][2]*X[3][3];
    Y[1][1] = X[0][2]*X[2][3]*X[3][0] - X[0][3]*X[2][2]*X[3][0] + X[0][3]*X[2][0]*X[3][2] - X[0][0]*X[2][3]*X[3][2] - X[0][2]*X[2][0]*X[3][3] + X[0][0]*X[2][2]*X[3][3];
    Y[1][2] = X[0][3]*X[1][2]*X[3][0] - X[0][2]*X[1][3]*X[3][0] - X[0][3]*X[1][0]*X[3][2] + X[0][0]*X[1][3]*X[3][2] + X[0][2]*X[1][0]*X[3][3] - X[0][0]*X[1][2]*X[3][3];
    Y[1][3] = X[0][2]*X[1][3]*X[2][0] - X[0][3]*X[1][2]*X[2][0] + X[0][3]*X[1][0]*X[2][2] - X[0][0]*X[1][3]*X[2][2] - X[0][2]*X[1][0]*X[2][3] + X[0][0]*X[1][2]*X[2][3];
    Y[2][0] = X[1][1]*X[2][3]*X[3][0] - X[1][3]*X[2][1]*X[3][0] + X[1][3]*X[2][0]*X[3][1] - X[1][0]*X[2][3]*X[3][1] - X[1][1]*X[2][0]*X[3][3] + X[1][0]*X[2][1]*X[3][3];
    Y[2][1] = X[0][3]*X[2][1]*X[3][0] - X[0][1]*X[2][3]*X[3][0] - X[0][3]*X[2][0]*X[3][1] + X[0][0]*X[2][3]*X[3][1] + X[0][1]*X[2][0]*X[3][3] - X[0][0]*X[2][1]*X[3][3];
    Y[2][2] = X[0][1]*X[1][3]*X[3][0] - X[0][3]*X[1][1]*X[3][0] + X[0][3]*X[1][0]*X[3][1] - X[0][0]*X[1][3]*X[3][1] - X[0][1]*X[1][0]*X[3][3] + X[0][0]*X[1][1]*X[3][3];
    Y[2][3] = X[0][3]*X[1][1]*X[2][0] - X[0][1]*X[1][3]*X[2][0] - X[0][3]*X[1][0]*X[2][1] + X[0][0]*X[1][3]*X[2][1] + X[0][1]*X[1][0]*X[2][3] - X[0][0]*X[1][1]*X[2][3];
    Y[3][0] = X[1][2]*X[2][1]*X[3][0] - X[1][1]*X[2][2]*X[3][0] - X[1][2]*X[2][0]*X[3][1] + X[1][0]*X[2][2]*X[3][1] + X[1][1]*X[2][0]*X[3][2] - X[1][0]*X[2][1]*X[3][2];
    Y[3][1] = X[0][1]*X[2][2]*X[3][0] - X[0][2]*X[2][1]*X[3][0] + X[0][2]*X[2][0]*X[3][1] - X[0][0]*X[2][2]*X[3][1] - X[0][1]*X[2][0]*X[3][2] + X[0][0]*X[2][1]*X[3][2];
    Y[3][2] = X[0][2]*X[1][1]*X[3][0] - X[0][1]*X[1][2]*X[3][0] - X[0][2]*X[1][0]*X[3][1] + X[0][0]*X[1][2]*X[3][1] + X[0][1]*X[1][0]*X[3][2] - X[0][0]*X[1][1]*X[3][2];
    Y[3][3] = X[0][1]*X[1][2]*X[2][0] - X[0][2]*X[1][1]*X[2][0] + X[0][2]*X[1][0]*X[2][1] - X[0][0]*X[1][2]*X[2][1] - X[0][1]*X[1][0]*X[2][2] + X[0][0]*X[1][1]*X[2][2];

    for (int i=0; i<4; i++)
        for (int j=0; j<4; j++)
            Y[i][j] = Y[i][j]/det;
}

void subtractVec(double *a, double *b, double *c, int s)
{
    for (int i=0; i<s; i++)
        c[i] = a[i]-b[i];
}

void negateVec(double *a, int s)
{
    for (int i=0; i<s; i++)
        a[i] = -a[i];
}

//s1,s2,s3: fastest to slowest
void sliceImageDouble(double *input, int s1, int s2, int s3, double *output, int indS1)
{
    for (int i=0; i<s3; i++)
        for (int j=0; j<s2; j++)
        {
            output[i*s2+j] = input[i*s2*s1+j*s1+indS1]*input[i*s2*s1+j*s1+s1-1];
        }
}

unsigned char quantizeDouble(double val, double minVal, double maxVal)
{
    return (val-minVal)*255.0/(maxVal-minVal);
}

//3D data, fastest to slowest
void quantizeImageDouble3D(double *input, unsigned char *output, int s0, int s1, int s2)
{
    double maxVal[4];
    maxVal[0] = maxVal[1] = maxVal[2] = maxVal[3] = -(1<<15);
    double minVal[4];
    minVal[0] = minVal[1] = minVal[2] = minVal[3] = ((1<<15) - 1);

    for (int i=0; i<s2; i++)
        for (int j=0; j<s1; j++)
            for (int k=0; k<s0; k++)
            {
                if (input[i*s1*s0+j*s0+k]>maxVal[k])
                    maxVal[k] = input[i*s1*s0+j*s0+k];
                if (input[i*s1*s0+j*s0+k]<minVal[k])
                    minVal[k] = input[i*s1*s0+j*s0+k];
            }
    for (int i=0; i<4; i++)
        printf("minmax %d = [%f,%f]\n",i,minVal[i],maxVal[i]);
    for (int i=0; i<s2; i++)
        for (int j=0; j<s1; j++)
            for (int k=0; k<s0; k++)
            {
                output[i*s1*s0+j*s0+k] = quantizeDouble(input[i*s1*s0+j*s0+k],minVal[k],maxVal[k]);
            }
}

template<class T>
void quantizeImage3D(T *input, unsigned char *output, int s0, int s1, int s2)
{
    double maxVal[4];
    maxVal[0] = maxVal[1] = maxVal[2] = maxVal[3] = -(1<<15);
    double minVal[4];
    minVal[0] = minVal[1] = minVal[2] = minVal[3] = ((1<<15) - 1);

    for (int i=0; i<s2; i++)
        for (int j=0; j<s1; j++)
            for (int k=0; k<s0; k++)
            {
                if (input[i*s1*s0+j*s0+k]>maxVal[k])
                    maxVal[k] = input[i*s1*s0+j*s0+k];
                if (input[i*s1*s0+j*s0+k]<minVal[k])
                    minVal[k] = input[i*s1*s0+j*s0+k];
            }
    for (int i=0; i<4; i++)
        printf("minmax %d = [%f,%f]\n",i,minVal[i],maxVal[i]);
    for (int i=0; i<s2; i++)
        for (int j=0; j<s1; j++)
            for (int k=0; k<s0; k++)
            {
                output[i*s1*s0+j*s0+k] = quantizeDouble(input[i*s1*s0+j*s0+k],minVal[k],maxVal[k]);
            }
}

void applyMask(unsigned char *input, int s0, int s1, int s2, int *mask, unsigned char *output)
{
    for (int i=0; i<s2; i++)
        for (int j=0; j<s1; j++)
            for (int k=0; k<s0; k++)
            {
                output[i*s1*s0+j*s0+k] = input[i*s1*s0+j*s0+k]*mask[i*s1+j];
            }
}

void removeChannel(unsigned char *input, int s0, int s1, int s2, int chan, unsigned char *output)
{
    memcpy(output,input,s0*s1*s2*sizeof(unsigned char));
    for (int i=0; i<s2; i++)
        for (int j=0; j<s1; j++)
                output[i*s1*s0+j*s0+chan] = 0;            
}
//---end of cuda_volume_rendering functions

template<class T>
void setPlane(T* image, int s1, int s2, int s3, T val, int s1i)
{
  for (int i=0; i<s3; i++)
    for (int j=0; j<s2; j++)
      image[i*s2*s1+j*s1+s1i] = val;
}

void transposeMat33(double X[][3], double Y[][3])
{
    for (int i=0; i<3; i++)
        for (int j=i; j<3; j++)
        {
            Y[i][j]=X[j][i];
            Y[j][i]=X[i][j];
        }
}


float lerp(float y0, float y1, float x0, float x, float x1)
{
  float alpha = (x-x0)/(x1-x0);
  return y0*(1-alpha)+alpha*y1;
}

float linearizeDepth(float depth, float zNear, float zFar)
{
    return (2.0 * zFar * zNear) / (zFar + zNear - depth * (zFar - zNear));
}

float linearizeDepthOrtho(float depth, float zNear, float zFar)
{
    return (depth*(zFar-zNear)+zFar+zNear)/2;
}



template<class T>
void saveImage(int width, int height, int nchan, T *data, char *name)
{
    TGAImage *img = new TGAImage(width,height);
    

    unsigned char* dataQuantized = new unsigned char[height*width*nchan];
    quantizeImage3D<T>(data,dataQuantized,nchan,width,height);

    Colour c;    
    for(int x=0; x<height; x++)
        for(int y=0; y<width; y++)
        {
            c.a = 255;
            c.b = c.g = c.r = 0;
            switch (nchan)
            {
              case 4:
                c.a = dataQuantized[x*width*nchan+y*nchan+3];
              case 3:
                c.b = dataQuantized[x*width*nchan+y*nchan+2];
              case 2:
                c.g = dataQuantized[x*width*nchan+y*nchan+1];
              case 1:
                c.r = dataQuantized[x*width*nchan+y*nchan];
            }                                        
            img->setPixel(c,x,y);
         }
    
    img->WriteImage(name);  
    delete img;
    delete[] dataQuantized;
}

template<class T>
void saveImageWithoutQuantizing(int width, int height, int nchan, T *data, char *name)
{
    TGAImage *img = new TGAImage(width,height);
    
    Colour c;    
    for(int x=0; x<height; x++)
        for(int y=0; y<width; y++)
        {
            c.a = 255;
            c.b = c.g = c.r = 0;
            switch (nchan)
            {
              case 4:
                c.a = data[x*width*nchan+y*nchan+3];
              case 3:
                c.b = data[x*width*nchan+y*nchan+2];
              case 2:
                c.g = data[x*width*nchan+y*nchan+1];
              case 1:
                c.r = data[x*width*nchan+y*nchan];
            }                                        
            img->setPixel(c,x,y);
        }
    
    img->WriteImage(name);  
    delete img;
}

//image1 and image2 should have same spatial size (except number of channels, i.e. fastest axis)
template <class T1, class T2>
void copyImageChannel(T1 *image1,int s10,int s11,int s12,int c1,T2 *image2,int s20,int c2)
{
  for (int i=0; i<s12; i++)
    for (int j=0; j<s11; j++)
    {
      int ind1 = i*s11*s10 + j*s10 + c1;
      int ind2 = i*s11*s20 + j*s20 + c2;
      image2[ind2] = image1[ind1];
    }
}

void render(Hale::Viewer *viewer){
  viewer->draw();
  viewer->bufferSwap();
}

int
main(int argc, const char **argv) {
  const char *me;
  char *err;
  hestOpt *hopt=NULL;
  hestParm *hparm;
  airArray *mop;

  char *name;
  char *texname1, *texname2;
  
  //double dir1[3]={1,0,0};
  //double dir2[3]={0,-1,0};
  double dir1[3],dir2[3];
  //double *dir1,*dir2;

  //tmp fixed track coords, and radius
  double track[3] = {366.653991263,89.6381792864,104.736646409};
  double trackhomo[4];
  trackhomo[0] = track[0];
  trackhomo[1] = track[1];
  trackhomo[2] = track[2];
  trackhomo[3] = 1;
  double trackw[4];
  double radius = 10;

//double *center;
  double center[3];
  //memcpy(center,track,sizeof(double)*3);

  int size[2];
  Nrrd *nin;
  char *outname;
  char inname[100];
  char *centername;
  double swidth, sstep; //width and step to take inside the slice  
  short *outdata;
  char outnameslice[100];



  /* boilerplate hest code */
  me = argv[0];
  mop = airMopNew();
  hparm = hestParmNew();
  airMopAdd(mop, hparm, (airMopper)hestParmFree, airMopAlways);
  /* setting up the command-line options */
  hparm->respFileEnable = AIR_TRUE;
  hparm->noArgsIsNoProblem = AIR_TRUE;

  //hestOptAdd(&hopt, "i", "nin", airTypeOther, 1, 1, &nin, "270.nrrd",
  //           "input volume to render", NULL, NULL, nrrdHestNrrd);
  //hestOptAdd(&hopt, "nseq", "start end", airTypeInt, 2, 2, nseq, "270 279",
  //           "start and end index of file names to process");  

  hestOptAdd(&hopt, "isize", "sx sy", airTypeInt, 2, 2, size, "200 200",
             "output image sizes");

  hestOptAdd(&hopt, "dir1", "x y z", airTypeDouble, 3, 3, dir1, "1 0 0",
             "first direction of the generated image");

  hestOptAdd(&hopt, "dir2", "x y z", airTypeDouble, 3, 3, dir2, "0 -1 0",
             "second direction of the generated image");

  hestOptAdd(&hopt, "swidth", "sw", airTypeDouble, 1, 1, &swidth, "1",
             "the width of the slice to cut");

  hestOptAdd(&hopt, "sstep", "ss", airTypeDouble, 1, 1, &sstep, "1",
             "the step of Maximum Intensity Projection through slice");  

  //hestOptAdd(&hopt, "center", "x y z", airTypeDouble, 3, 3, center, "366.653991263 89.6381792864 104.736646409",
  //           "center of the generated image");
  hestOptAdd(&hopt, "i", "name", airTypeString, 1, 1, &centername, "coord_newtrack_pioneer", "name of files centaining centers");

  hestOptAdd(&hopt, "o", "name", airTypeString, 1, 1, &outname, "cpr.nrrd", "name of output image");

  hestParseOrDie(hopt, argc-1, argv+1, hparm,
                 me, "demo program", AIR_TRUE, AIR_TRUE, AIR_TRUE);
  airMopAdd(mop, hopt, (airMopper)hestOptFree, airMopAlways);
  airMopAdd(mop, hopt, (airMopper)hestParseFree, airMopAlways);

  /* Compute threshold (isovalue) */
  cout<<"After TEEM processing of input arguments"<<endl;

  int countline = 0;
  string line;
  ifstream infile(centername);
  int *arr_nameid;
  double *arr_center;

  while (std::getline(infile, line))
  {
    ++countline;    
  }

  infile.clear();
  infile.seekg(0, ios::beg);

  arr_nameid = new int[countline];
  arr_center = new double[countline*3];
  for (int i=0; i<countline; i++)
  {
    infile >> arr_nameid[i];
    infile >> arr_center[i*3];
    infile >> arr_center[i*3+1];
    infile >> arr_center[i*3+2];
  }
  infile.close();
  cout<<"Initialized countline = "<<countline<<endl;

  double thresdis = 1;
  vector<double> vcenter;

  vcenter.push_back(arr_center[0]);
  vcenter.push_back(arr_center[1]);
  vcenter.push_back(arr_center[2]);


  for (int i=1; i<countline; i++)
  {
    int countv = vcenter.size();
    if (diss2P(vcenter[countv-3],vcenter[countv-2],vcenter[countv-1],arr_center[i*3+0],arr_center[i*3+1],arr_center[i*3+2])<thresdis)
    {
      continue;
    }
    else
    {
      vcenter.push_back(arr_center[i*3+0]);
      vcenter.push_back(arr_center[i*3+1]);
      vcenter.push_back(arr_center[i*3+2]);
      countv = vcenter.size()/3;
      arr_nameid[countv-1] = arr_nameid[i];
    }
  }
  printf("after correcting input\n");
  countline = vcenter.size()/3;
  memcpy(arr_center,vcenter.data(),sizeof(double)*countline*3);

  outdata = new short[size[0]*size[1]*countline];

  cout<<"Initialized outdata"<<endl;

  int curnameind;
  
  float* filemem0 = NULL;
  float* filemem1 = NULL;
  int initalized = 0;
  double *imageDouble = NULL;
  int *d_dim;
  double *d_dir1;
  double *d_dir2;
  double *d_imageDouble;
  int *d_size;
  double *d_center;
  int count = 0;

  nin = nrrdNew();

  Nrrd *ndblpng = nrrdNew();

  float camfr[3], camat[3], camup[3], camnc, camfc, camFOV;
  int camortho;
  unsigned int camsize[2];
  camfr[0] = arr_center[0];
  camfr[1] = arr_center[1];
  camfr[2] = arr_center[2]-50;
  camat[0] = arr_center[0];
  camat[1] = arr_center[1];
  camat[2] = arr_center[2];
  camup[0] = 1;
  camup[1] = 0;
  camup[2] = 0;
  camnc = -500;
  camfc = 500;
  camFOV = 170;
  camortho = 1;
  camsize[0] = 500;
  camsize[1] = 500;

  Hale::init();
  Hale::Scene scene;
  /* then create viewer (in order to create the OpenGL context) */
  Hale::Viewer viewer(camsize[0], camsize[1], "Iso", &scene);
  viewer.lightDir(glm::vec3(-1.0f, 1.0f, 3.0f));
  viewer.camera.init(glm::vec3(camfr[0], camfr[1], camfr[2]),
                     glm::vec3(camat[0], camat[1], camat[2]),
                     glm::vec3(camup[0], camup[1], camup[2]),
                     camFOV, (float)camsize[0]/camsize[1],
                     camnc, camfc, camortho);
  viewer.refreshCB((Hale::ViewerRefresher)render);
  viewer.refreshData(&viewer);
  viewer.current();

  printf("Initialized viewer\n");

  Hale::Program *newprog = new Hale::Program("tex-vert-cpr.glsl","texdemo-frag.glsl");
  newprog->compile();
  newprog->bindAttribute(Hale::vertAttrIdxXYZW, "positionVA");
  newprog->bindAttribute(Hale::vertAttrIdxRGBA, "colorVA");
  newprog->bindAttribute(Hale::vertAttrIdxNorm, "normalVA");
  newprog->bindAttribute(Hale::vertAttrIdxTex2, "tex2VA");
  newprog->link();    

  //adding some points outside of the valid convolution range
  int pointind[3];
  pointind[0] = 0;
  pointind[1] = countline-1;
  pointind[2] = countline-2;
  double spherescale = 0.4;
  for (int i=0; i<3; i++)
  {
    limnPolyData *lpld2 = limnPolyDataNew();
    limnPolyDataIcoSphere(lpld2, 1 << limnPolyDataInfoNorm, 3);

    Hale::Polydata *hpld2 = new Hale::Polydata(lpld2, true,
                         Hale::ProgramLib(Hale::preprogramAmbDiffSolid),
                         "IcoSphere");
    hpld2->colorSolid(lerp(0,1,0,pointind[i],countline-1),lerp(1,0,0,pointind[i],countline-1),0.5);
    
    glm::mat4 fmat2 = glm::mat4();
    
    fmat2[0][0] = spherescale;
    fmat2[1][1] = spherescale;
    fmat2[2][2] = spherescale;
    fmat2[3][0] = arr_center[pointind[i]*3+0];
    fmat2[3][1] = arr_center[pointind[i]*3+1];
    fmat2[3][2] = arr_center[pointind[i]*3+2];
    fmat2[3][3] = 1;
    

    hpld2->model(fmat2);    

    scene.add(hpld2);   
  }

  for (count = 1; count<countline-2; count++)
  {
    //infile >> curnameind;
    //infile >> center[0] >> center[1] >> center[2];
    curnameind = arr_nameid[count];
    center[0] = arr_center[count*3];
    center[1] = arr_center[count*3+1];
    center[2] = arr_center[count*3+2];
    
    double FT[3];
    double FN[3],FB[3];
    double dr[3],ddr[3];
    for (int i=0; i<3; i++)
      dr[i] = cubicFilter_G<double>(0, arr_center[(count-1)*3+i], arr_center[(count)*3+i], arr_center[(count+1)*3+i], arr_center[(count+2)*3+i]);
    //dr[1] = cubicFilter_G<double>(0, arr_center[(count-1)*3+1], arr_center[(count)*3+1], arr_center[(count+1)*3+1], arr_center[(count+2)*3+1]);
    //dr[2] = cubicFilter_G<double>(0, arr_center[(count-1)*3+2], arr_center[(count)*3+2], arr_center[(count+1)*3+2], arr_center[(count+2)*3+2]);
    for (int i=0; i<3; i++)
      ddr[i] = cubicFilter_GG<double>(0, arr_center[(count-1)*3+i], arr_center[(count)*3+i], arr_center[(count+1)*3+i], arr_center[(count+2)*3+i]);

    normalize(dr,3);
    normalize(ddr,3);

    memcpy(FT,dr,sizeof(double)*3);
    double crossddrdr[3];
    cross(ddr,dr,crossddrdr);
    cross(dr,crossddrdr,FN);
    normalize(FN,3);
    cross(FT,FN,FB);
    memcpy(dir1,FN,sizeof(double)*3);
    memcpy(dir2,FB,sizeof(double)*3);
    printf("N = %f %f %f, B = %f %f %f, T = %f %f %f, dotNB = %f, dotNT = %f, dotBT = %f\n",FN[0],FN[1],FN[2],FB[0],FB[1],FB[2],FT[0],FT[1],FT[2],
      dotProduct(FN,FB,3),dotProduct(FN,FT,3),dotProduct(FB,FT,3));

    limnPolyData *lpld = limnPolyDataNew();
    limnPolyDataSquare(lpld, 1 << limnPolyDataInfoNorm | 1 << limnPolyDataInfoTex2);

    printf("after initializing lpld\n");
    
    //Hale::Polydata *hpld = new Hale::Polydata(lpld, true,
    //                     Hale::ProgramLib(Hale::preprogramAmbDiffSolid),
    //                     "square");
    Hale::Polydata *hpld = new Hale::Polydata(lpld, true,
                         NULL,
                         "square");
    hpld->program(newprog);
    //hpld->colorSolid(lerp(0,1,0,count,countline-1),lerp(1,0,0,count,countline-1),0.5);
    //printf("after setting color for hpld\n");
    
    glm::mat4 tmat = glm::mat4();
    
    tmat[0][0] = FN[0];
    tmat[0][1] = FN[1];
    tmat[0][2] = FN[2];
    tmat[0][3] = 0;
    tmat[1][0] = FB[0];
    tmat[1][1] = FB[1];
    tmat[1][2] = FB[2];
    tmat[1][3] = 0;
    tmat[2][0] = FT[0];
    tmat[2][1] = FT[1];
    tmat[2][2] = FT[2];
    tmat[2][3] = 0;
    tmat[3][0] = center[0];
    tmat[3][1] = center[1];
    tmat[3][2] = center[2];
    tmat[3][3] = 1;
    
    glm::mat4 smat = glm::mat4();
    smat[0][0] = 2;
    smat[1][1] = 2;
    glm::mat4 fmat = tmat*smat;

    hpld->model(fmat);    


    
//add a sphere
    limnPolyData *lpld2 = limnPolyDataNew();
    limnPolyDataIcoSphere(lpld2, 1 << limnPolyDataInfoNorm, 3);

    Hale::Polydata *hpld2 = new Hale::Polydata(lpld2, true,
                         Hale::ProgramLib(Hale::preprogramAmbDiffSolid),
                         "IcoSphere");
    hpld2->colorSolid(lerp(0,1,0,count,countline-1),lerp(1,0,0,count,countline-1),0.5);
    
    glm::mat4 fmat2 = glm::mat4();
    
    fmat2[0][0] = spherescale;
    fmat2[1][1] = spherescale;
    fmat2[2][2] = spherescale;
    fmat2[3][0] = center[0];
    fmat2[3][1] = center[1];
    fmat2[3][2] = center[2];
    fmat2[3][3] = 1;
    

    hpld2->model(fmat2);    

    scene.add(hpld2);    
    
    printf("after adding hpld to scene\n");

    printf("added lpld\n");


    cout<<"Before read in file, with curnameind = "<<curnameind<<", center = "<<center[0]<<" "<<center[1]<<" "<<center[2]<<endl;
    sprintf(inname,"/media/trihuynh/781B8CE3469A7908/scivisdata/%d.nrrd",curnameind);
    cout<<"inname = "<<inname<<endl;

    if (nrrdLoad(nin, inname, NULL)) {
      err = biffGetDone(NRRD);
      fprintf(stderr, "%s: trouble reading \"%s\":\n%s", me, inname, err);
      free(err);
      return;
    }

    cout<<"read file "<<inname<<endl;
    unsigned int pixSize;
    hipChannelFormatDesc channelDesc;
    pixSize = sizeof(float);
    channelDesc = hipCreateChannelDesc<float>();

    if (3 != nin->dim && 3 != nin->spaceDim) {
        fprintf(stderr, "%s: need 3D array in 3D space, (not %uD in %uD)\n",
        argv[0], nin->dim, nin->spaceDim);
        airMopError(mop); exit(1);
    }

    double mat_trans[4][4];

    mat_trans[3][0] = mat_trans[3][1] = mat_trans[3][2] = 0;
    mat_trans[3][3] = 1;

    int dim[4];
    if (nin->dim == 3)
    {
        dim[0] = 1;
        dim[1] = nin->axis[0].size;
        dim[2] = nin->axis[1].size;
        dim[3] = nin->axis[2].size;
        for (int i=0; i<3; i++) {
            for (int j=0; j<3; j++) {
                /* for 2-channel data; this "i" should be "i+1" */
                mat_trans[j][i] = nin->axis[i].spaceDirection[j];
            }
            mat_trans[i][3] = nin->spaceOrigin[i];
        }
    }
    else //4-channel
    {
        dim[0] = nin->axis[0].size;
        dim[1] = nin->axis[1].size;
        dim[2] = nin->axis[2].size;
        dim[3] = nin->axis[3].size;
        for (int i=0; i<3; i++) {
            for (int j=0; j<3; j++) {
                /* for 2-channel data; this "i" should be "i+1" */
                mat_trans[j][i] = nin->axis[i+1].spaceDirection[j];
            }
            mat_trans[i][3] = nin->spaceOrigin[i];
        }
    }
    int channel = 1;
    //int filesize = dim[0]*dim[1]*dim[2]*dim[3]*pixSize;

    if (!initalized)
    {
      filemem0 = new float[dim[1]*dim[2]*dim[3]];
      filemem1 = new float[dim[1]*dim[2]*dim[3]];
    }

    //filemem = (char*)nin->data;
    for (int i=0; i<dim[1]*dim[2]*dim[3]; i++)
    {
        filemem0[i] = ((short*)nin->data)[i*2];
        filemem1[i] = ((short*)nin->data)[i*2+1];
    }

    double mat_trans_inv[4][4];
    invertMat44(mat_trans,mat_trans_inv);
   //tex3D stuff
    const hipExtent volumeSize = make_hipExtent(dim[1], dim[2], dim[3]);

    //hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    if (!initalized)
    {
      hipMalloc3DArray(&d_volumeArray0, &channelDesc, volumeSize);
      hipMalloc3DArray(&d_volumeArray1, &channelDesc, volumeSize);
    }

    // --- Copy data to 3D array (host to device)
    hipMemcpy3DParms copyParams1 = {0};
    copyParams1.srcPtr   = make_hipPitchedPtr((void*)filemem1, volumeSize.width*pixSize, volumeSize.width, volumeSize.height);
    copyParams1.dstArray = d_volumeArray1;
    copyParams1.extent   = volumeSize;
    copyParams1.kind     = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams1);

    hipMemcpy3DParms copyParams0 = {0};
    copyParams0.srcPtr   = make_hipPitchedPtr((void*)filemem0, volumeSize.width*pixSize, volumeSize.width, volumeSize.height);
    copyParams0.dstArray = d_volumeArray0;
    copyParams0.extent   = volumeSize;
    copyParams0.kind     = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams0);
    // --- Set texture parameters
    tex1.normalized = false;                      // access with normalized texture coordinates
    tex1.filterMode = hipFilterModeLinear;      // linear interpolation
    /*
    tex1.addressMode[0] = hipAddressModeWrap;   // wrap texture coordinates
    tex1.addressMode[1] = hipAddressModeWrap;
    tex1.addressMode[2] = hipAddressModeWrap;
    */
    tex1.addressMode[0] = hipAddressModeBorder;   // wrap texture coordinates
    tex1.addressMode[1] = hipAddressModeBorder;
    tex1.addressMode[2] = hipAddressModeBorder;


    tex0.normalized = false;                      // access with normalized texture coordinates
    tex0.filterMode = hipFilterModeLinear;      // linear interpolation
    /*
    tex0.addressMode[0] = hipAddressModeWrap;   // wrap texture coordinates
    tex0.addressMode[1] = hipAddressModeWrap;
    tex0.addressMode[2] = hipAddressModeWrap;
    */
    tex0.addressMode[0] = hipAddressModeBorder;   // wrap texture coordinates
    tex0.addressMode[1] = hipAddressModeBorder;
    tex0.addressMode[2] = hipAddressModeBorder;
    // --- Bind array to 3D texture
    hipBindTextureToArray(tex1, d_volumeArray1, channelDesc);
    hipBindTextureToArray(tex0, d_volumeArray0, channelDesc);
    //-----------

    int nOutChannel = 4;

    if (!initalized)
    {
      imageDouble = new double[size[0]*size[1]*nOutChannel];

      hipMalloc(&d_dim, sizeof(dim));
      hipMemcpy(d_dim, dim, 4*sizeof(int), hipMemcpyHostToDevice);

      hipMalloc(&d_dir1, sizeof(dir1));
      hipMemcpy(d_dir1, dir1, 3*sizeof(double), hipMemcpyHostToDevice);

      hipMalloc(&d_dir2, sizeof(dir2));
      hipMemcpy(d_dir2, dir2, 3*sizeof(double), hipMemcpyHostToDevice);

      hipMalloc(&d_imageDouble,sizeof(double)*size[0]*size[1]*nOutChannel);

      hipMalloc(&d_size,2*sizeof(int));
      hipMemcpy(d_size,size,2*sizeof(int), hipMemcpyHostToDevice);

      hipMalloc(&d_center,3*sizeof(double));
    }

    hipMemcpy(d_center,center,3*sizeof(double), hipMemcpyHostToDevice);


    int numThread1D = 16;
    dim3 threadsPerBlock(numThread1D,numThread1D);
    dim3 numBlocks((size[0]+numThread1D-1)/numThread1D,(size[1]+numThread1D-1)/numThread1D);

    kernel_cpr<<<numBlocks,threadsPerBlock>>>(d_dim, d_size, d_center, d_dir1, d_dir2, swidth, sstep, nOutChannel, d_imageDouble);

    hipError_t errCu = hipGetLastError();
    if (errCu != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(errCu));

    errCu = hipDeviceSynchronize();
    if (errCu != hipSuccess) 
        printf("Error Sync: %s\n", hipGetErrorString(errCu));

    hipMemcpy(imageDouble, d_imageDouble, sizeof(double)*size[0]*size[1]*nOutChannel, hipMemcpyDeviceToHost);

    short width = size[0];
    short height = size[1];

    copyImageChannel<double,short>(imageDouble,4,size[0],size[1],0,outdata+count*size[0]*size[1],1,0);

    unsigned char *imageQuantized = new unsigned char[size[0]*size[1]*4];
    quantizeImageDouble3D(imageDouble,imageQuantized,4,size[0],size[1]);    
    setPlane<unsigned char>(imageQuantized, 4, size[0], size[1], 255, 3);

    hpld->setTexture((char*)"myTextureSampler",(unsigned char *)imageQuantized,size[0],size[1],4);
    scene.add(hpld);
    
    drawCircle(imageQuantized,4,size[0],size[1],1,size[0]/2,size[1]/2,20);
//end of cuda_rendering

    //sprintf(outnameslice,"cpr_seq_%d.tga",curnameind);
    //saveImageWithoutQuantizing<unsigned char>(size[0],size[1],4,imageQuantized,outnameslice);

    initalized = 1;
    //count++;
    sprintf(outnameslice,"cpr_seq_%d.png",curnameind);
    if (nrrdWrap_va(ndblpng, imageQuantized, nrrdTypeUChar, 3, 4, width, height)
      || nrrdSave(outnameslice, ndblpng, NULL)
          ) {
      char *err = biffGetDone(NRRD);
      printf("%s: couldn't save output:\n%s", argv[0], err);
      free(err); nrrdNix(ndblpng);
      exit(1);
      }
  }

  cout<<"Before allocating output nrrd"<<endl;  
  Nrrd *ndbl = nrrdNew();

  cout<<"Before saving output nrrd"<<endl;
  if (nrrdWrap_va(ndbl, outdata, nrrdTypeShort, 3, size[0], size[1], countline)
        || nrrdSave(outname,ndbl,NULL)
        ) 
  {
    char *err = biffGetDone(NRRD);
    printf("%s: couldn't save output:\n%s", argv[0], err);
    free(err); nrrdNix(ndbl);
    exit(1);
  }

  cout<<"After saving output nrrd"<<endl;
  scene.drawInit();
  printf("after scene.drawInit()\n");
  render(&viewer);
  printf("after render(&viewer)\n");
  while(!Hale::finishing){
    glfwWaitEvents();
    render(&viewer);
  }

  /* clean exit; all okay */
  Hale::done();
 
  airMopOkay(mop);

  return 0;
}
